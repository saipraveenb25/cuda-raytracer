#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "cuda_image.h"
//#include "noise.h"
//#include "sceneLoader.h"
//#include "util.h"
//#include "cycleTimer.h"
#define SCAN_BLOCK_DIM 1024  // needed by sharedMemExclusiveScan implementation
#include "exclusiveScan.cu_inl"

#include "cuda_util.h"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

namespace cutracer {

float3 v2f3(Vector3D v) {
    return make_float3(v.x, v.y, v.z);
}

Vector3D f32v(float3 f) {
    return Vector3D(f.x, f.y, f.z);
}
/*struct CuRay {
    float3 o;
    float3 d;
    float3 importance; // Importance weight for this ray at this point.
    float3 light; // Direct light that is passing through this point so far.
    float3 lightImportance; // Non-zero for a light-intersection connection.
    float2 ss;  // Screen Space coordinates for update.
    float maxt; // Maximum length.
    int sid; // Sample ID.
};

struct CuTriangle {
    float3 a;
    float3 b;
    float3 c;

    // TODO:  Check if necessary. Edge normals.
    float3 n1;
    float3 n2;
    float3 n3;

    int bsdf; // BSDF index.
    int emit; // Emitter index.
};

struct CuEmitter {
    float3 radiance;
    float3 position;
    float3 direction;
    float3 dim_x;
    float3 dim_y;
    float area;
};

struct CuBSDF{
    int fn; // 0 - diffuse, 1 - specular.
    float3 albedo;  // For diffuse.
    float nu;       // For specular.
};

struct CuBVHSubTree {
    
    uint64_t outlets[16];
    
    uint64_t start;
    uint64_t range;
    
    float3 minl[16];
    float3 maxl[16];
};

struct CuIntersection {
    float3 pt;
    float3 n;
    float3 light;
    float3 wi;
    float3 importance;
    float t;
    float maxT;
    float2 ss;
    int sid;
    int bsdf;
};*/ 
void init_camera(Collada::CameraInfo &cameraInfo,
                              const Matrix4x4 &transform) {
  //camera.configure(cameraInfo, screenW, screenH);
  //canonicalCamera.configure(cameraInfo, screenW, screenH);
  //set_projection_matrix();
}

DynamicScene::SceneLight *init_light(Collada::LightInfo &light,
                                                  const Matrix4x4 &transform) {
  switch (light.light_type) {
    case Collada::LightType::NONE:
      break;
    case Collada::LightType::AMBIENT:
      return new DynamicScene::AmbientLight(light);
    case Collada::LightType::DIRECTIONAL:
      return new DynamicScene::DirectionalLight(light, transform);
    case Collada::LightType::AREA:
      return new DynamicScene::AreaLight(light, transform);
    case Collada::LightType::POINT:
      return new DynamicScene::PointLight(light, transform);
    case Collada::LightType::SPOT:
      return new DynamicScene::SpotLight(light, transform);
    default:
      break;
  }
  return nullptr;
}

/**
 * The transform is assumed to be composed of translation, rotation, and
 * scaling, where the scaling is uniform across the three dimensions; these
 * assumptions are necessary to ensure the sphere is still spherical. Rotation
 * is ignored since it's a sphere, translation is determined by transforming the
 * origin, and scaling is determined by transforming an arbitrary unit vector.
 */
DynamicScene::SceneObject *init_sphere(
    Collada::SphereInfo &sphere, const Matrix4x4 &transform) {
  const Vector3D &position = (transform * Vector4D(0, 0, 0, 1)).projectTo3D();
  double scale = (transform * Vector4D(1, 0, 0, 0)).to3D().norm();
  return new DynamicScene::Sphere(sphere, position, scale);
}

DynamicScene::SceneObject *init_polymesh(
    Collada::PolymeshInfo &polymesh, const Matrix4x4 &transform) {
  return new DynamicScene::Mesh(polymesh, transform);
}

struct GlobalConstants {

    // Image data.
    int imageWidth;
    int imageHeight;
    float4* ssImageData; // Super sampled image data.
    float4* imageData; // Final image data.
    int sampleCount;

    // Camera data.
    float3 c_origin;
    float3 c_lookAt;
    float3 c_up;
    float3 c_left;

    // Ray queues: SxR (S=Number of subtrees, R=Max number of rays per queue) 
    // (Gigantic ~32M entries)
    CuRay* queues;

    // Queue counts. Initialized to 0.
    uint* qCounts;

    // Ray intersection buffers. (Large 1K-32K entries)
    CuIntersection* intersections;

    // Triangle list. (Huge 1000-1000000 entries)
    CuTriangle* triangles;

   // Emitter list. (Tiny 1-2 entries)
    CuEmitter* emitters;

    // BSDF list. (Small ~10 entries).
    CuBSDF* bsdfs;

    // BVHNode list
    // (Large 10-4000 entries)
    CuBVHSubTree* bvhSubTrees;

    int* levelIndices;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

__device__ __inline__ void cudaswap(float& a, float& b) {
    float temp = a;
    a = b;
    b = temp;
}

__device__ __inline__ float intersectBBox(float3 o, float3 d, float3 min, float3 max) {
    float tmin = (min.x - o.x) / d.x; 
    float tmax = (max.x - o.x) / d.x; 

    if (tmin > tmax) cudaswap(tmin, tmax); 

    float tymin = (min.y - o.y) / d.y; 
    float tymax = (max.y - o.y) / d.y; 

    if (tymin > tymax) cudaswap(tymin, tymax); 

    if ((tmin > tymax) || (tymin > tmax)) 
        return -1; 

    if (tymin > tmin) 
        tmin = tymin; 

    if (tymax < tmax) 
        tmax = tymax; 

    float tzmin = (min.z - o.z) / d.z; 
    float tzmax = (max.z - o.z) / d.z; 

    if (tzmin > tzmax) cudaswap(tzmin, tzmax); 

    if ((tmin > tzmax) || (tzmin > tmax)) 
        return -1;

    if (tzmin > tmin) 
        tmin = tzmin; 

    if (tzmax < tmax) 
        tmax = tzmax; 

    return tmin;

}

/*__device__ __inline__ float3 crossProduct(float3 a, float3 b) {
  return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x - b.z, a.x * b.y - a.y * b.x);
  }

  __device__ __inline__ float dotProduct(float3 a, float3 b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
  }*/

__device__ __inline__ float intersectRayTriangle(float3 v0, float3 v1, float3 v2, float3 orig, float3 dir) {
    //float e1[3],e2[3],h[3],s[3],q[3];
    //float a,f,u,v;
    //vector(e1,v1,v0);
    //vector(e2,v2,v0);
    // compute plane's normal
    float3 v0v1 = v1 - v0; 
    float3 v0v2 = v2 - v0; 
    // no need to normalize
    float3 N = cross(v0v1, v0v2); // N 
    float area2 = length(N);

    // Step 1: finding P

    // check if ray and plane are parallel ?
    float NdotRayDirection = dot(N, dir); 
    if (abs(NdotRayDirection) < 1e-5) // almost 0 
        return -1; // they are parallel so they don't intersect ! 

    // compute d parameter using equation 2
    float d = dot(N, v0); 

    // compute t (equation 3)
    float t = (dot(N, orig) + d) / NdotRayDirection; 

    // check if the triangle is in behind the ray
    if (t < 0) return -1; // the triangle is behind 

    // compute the intersection point using equation 1
    float3 P = orig + t * dir; 

    // Step 2: inside-outside test
    float3 C; // vector perpendicular to triangle's plane 

    // edge 0
    float3 edge0 = v1 - v0;
    float3 vp0 = P - v0;
    C = cross(edge0, vp0);
    if (dot(N, C) < 0) return -1; // P is on the right side 

    // edge 1
    float3 edge1 = v2 - v1; 
    float3 vp1 = P - v1; 
    C = cross(edge1, vp1); 
    if (dot(N, C) < 0)  return -1; // P is on the right side 

    // edge 2
    float3 edge2 = v0 - v2; 
    float3 vp2 = P - v2; 
    C = cross(edge2, vp2); 
    if (dot(N, C) < 0) return -1; // P is on the right side; 

    return t; // this ray hits the triangle 
}

__global__ void kernelClearBuffers(float3* positions, float* radii, float3* colors, int* queues) {
    //int idx = threadIdx.x;
    //int block = blockIdx.x;
    //int layer = blockIdx.y;

    //positions[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = make_float3(0.f,0.f,0.f);
    //radii[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = 0.f;
    //colors[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = make_float3(0.f,0.f,0.f);
    //queues[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = 0;
}
// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}



// PT functions.

// Generate camera rays and insert into queue.
__global__ void kernelPrimaryRays( ) {

    // For each pixel
    // For each sample per pixel
    // Create ray and set to queue at the falling icicomputed offset.

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    //int raycount = cuConstRendererParams.rayCount;


    int sampleCount = cuConstRendererParams.sampleCount;

    for(int i = 0; i < sampleCount; i++) {
        int destIndex = i + (imageX * height + imageY) * sampleCount;

        float2 sample = make_float2(0.5f, 0.5f);

        float xs = imageX + sample.x;
        float ys = imageY + sample.y;

        float2 ss = make_float2(xs, ys);

        float a = (ss.x / width) - 0.5;
        float b = (ss.y / height) - 0.5;
        float c = sqrt(1 - (a*a + b*b));

        float px = a * cuConstRendererParams.c_left.x + b * cuConstRendererParams.c_up.x + c * cuConstRendererParams.c_lookAt.x;
        float py = a * cuConstRendererParams.c_left.y + b * cuConstRendererParams.c_up.y + c * cuConstRendererParams.c_lookAt.y;
        float pz = a * cuConstRendererParams.c_left.z + b * cuConstRendererParams.c_up.z + c * cuConstRendererParams.c_lookAt.z;

        CuRay *r = &cuConstRendererParams.queues[destIndex];
        r->o = cuConstRendererParams.c_origin;
        r->d = make_float3(px, py, pz);
        r->importance = make_float3(1, 1, 1);
        r->lightImportance = make_float3(0, 0, 0);
        r->light = make_float3(0, 0, 0);
        r->maxT = INFINITY; // TODO: Put inf.
        r->ss = ss;
        r->sid = i;
    }

}

// Generate direct light rays from intersections.
__global__ void kernelDirectLightRays() {
    // For each element in intersection. (Map each intersection to a thread).
    // For each light
    // Create a ray from light to intersection.
    // Add ray to queue.

    int iid = blockIdx.x * blockDim.x + threadIdx.x;

    CuIntersection *its = &cuConstRendererParams.intersections[iid];

    CuEmitter *e = &cuConstRendererParams.emitters[0];

    // Generate the sample.
    float sampleX;
    float sampleY;

    //float lX = (sampleX * e->minl.x + (1 - sampleX) * e->maxl.x);
    //float lY = (sampleY * e->minl.y + (1 - sampleY) * e->maxl.y);
    //float lZ = e->lmin.z; // Assume e->lmin.z = e->lmax.z;
    //Vector2D sample = sampler.get_sample() - Vector2D(0.5f, 0.5f);
    CuRay *r = &cuConstRendererParams.queues[iid];
    
    float3 d = e->position + sampleX * e->dim_x + sampleY * e->dim_y - its->pt;
    float cosTheta = dot(d, e->direction);
    float sqDist = dot(d,d);
    float dist = sqrt(sqDist);
    r->d = d / dist;
    float distToLight = dist;
    float pdf = sqDist / (e->area * abs(cosTheta));
    float fpdf = abs(dot(its->n, r->d))/ pdf;
    r->lightImportance = its->importance * make_float3(fpdf, fpdf, fpdf);
    r->maxT = distToLight;
    r->importance = its->importance;
    r->sid = its->sid;
    r->light = its->light;

    //return cosTheta < 0 ? radiance : Spectrum();


    // Conenct light to point.

    /*float dX = -its->pt.x + lX;
    float dY = -its->pt.y + lY;
    float dZ = -its->pt.z + lZ;
    

    float l = sqrt(dX * dX + dY * dY + dZ * dZ);


    float3 d = make_float3(dX / l, dY / l, dZ / l);
    float3 o = its->pt;
    r->d = d;
    r->o = o;
    //float3 importance;// copy total importance.
    //float3 lightImportance; // Compute (n.l)/(p - l)^2

    //float3 light; // copy light.

    r->light = its->light;
    r->importance = its->importance;
    float invLight = 1.0f / dot(r->d - its->pt, r->d - its->pt);

    //r->lightImportance = its->importance * e->radiance * e->area * dot(e->direction, r->d) * dot(its->n, r->d) * invLight;// Compute (n.l) / (p - l)^2
    //r->maxT = sqrt(dot(r->d - its->pt, r->d - its->pt)) - 0.01f;
    //r->lightImportance = make_float3(xf, xf, xf);*/
    //r->sid = its->sid;
}
#define BSDF_DIFFUSE_MULTIPLIER 1.0
// Generate secondary rays from the given intersections.
__global__ void kernelProcessIntersections( ) {

    // For each element in intersection.
    // Check BSDF.
    // If 0 (Diffuse):
    // Randomly sample each intersection.
    // If 1 (Specular):
    // Find reflected ray.

    // Create new ray.
    // Compute importance for this ray.
    // Add to ray list at the same space as the intersection.

    int iid = blockIdx.x * blockDim.x + threadIdx.x;
    CuIntersection *its = &cuConstRendererParams.intersections[iid];
    CuRay *r = &cuConstRendererParams.queues[0];

    // Compute a random sample. Hemispherical Random Sample.
    float sampleX;
    float sampleY;
    float sampleZ;

    float3 n = its->n;
    float3 dpdu; //TODO:Compute;
    float3 dpdv; //TODO:Compute;

    float dX = n.x * sampleZ + sampleX * dpdu.x + sampleY * dpdv.x;
    float dY = n.y * sampleZ + sampleX * dpdu.y + sampleY * dpdv.y;
    float dZ = n.z * sampleZ + sampleX * dpdu.z + sampleY * dpdv.z;

    float3 d = make_float3(dX, dY, dZ);
    float3 o = its->pt;
    r->d = d;
    r->o = o;

    int bsdfID = its->bsdf;
    CuBSDF *bsdf = &cuConstRendererParams.bsdfs[bsdfID];

    if(bsdf->fn == 0) {
        r->importance = its->importance * dot(r->d, its->n) * bsdf->albedo * BSDF_DIFFUSE_MULTIPLIER; // TODO: Compute with BSDF.
        r->light = its->light;
        r->lightImportance = make_float3(0, 0, 0);
        r->maxT = INFINITY;
        r->sid = its->sid;
    } else if(bsdf->fn == 1){
        // TODO: Implement specular stuff.
        r->importance = its->importance * dot(r->d, its->n); // TODO: Compute with BSDF.
        r->light = its->light;
        r->lightImportance = make_float3(0, 0, 0);
        r->maxT = INFINITY;
        r->sid = its->sid;
    }
}

__global__ void kernelUpdateSSImage( ) {
    // For each element in intersection list.
    // Update the its.ss pixels using a reconstruction filter into
    // imageData.

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;
    int sampleCount = cuConstRendererParams.sampleCount;

    int iid = blockIdx.x * blockDim.x + threadIdx.x;

    CuIntersection *its = &cuConstRendererParams.intersections[iid];

    int x = static_cast<int>(its->ss.x);
    int y = static_cast<int>(its->ss.y);

    int sid = its->sid;

    float4 *fx = &cuConstRendererParams.imageData[((y * height + x) * sampleCount + sid)];
    *fx = make_float4(its->light.x, its->light.y, its->light.z, 1.0);

}

// Box Filter.
// Soon change to Gaussian.
__global__ void kernelReconstructImage( ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int scount = cuConstRendererParams.sampleCount;

    float4 color = make_float4(0, 0, 0, 0);
    for(int i = 0; i < scount; i++) {
        float4 localColor;
        localColor = cuConstRendererParams.ssImageData[idx * scount + i];
        /*localColor.x = cuConstRendererParams.ssImageData[idx * scount + i].x;
        localColor.y = cuConstRendererParams.ssImageData[idx * scount + i].y;
        localColor.z = cuConstRendererParams.ssImageData[idx * scount + i].y;
        localColor.w = cuConstRendererParams.ssImageData[idx * scount + i].w;*/
        

        /*color.x += localColor[0] / scount;
        color.y += localColor[1] / scount;
        color.z += localColor[2] / scount;
        color.w += localColor[3] / scount;*/

        color += localColor / scount;
        
    }

    cuConstRendererParams.imageData[idx] = color;
}

// Intersection functions
// Performs ray intersect on a single node.
__device__ void rayIntersectSingle(int snode, int index) {

    // Mapping: Each block takes a subset of rays.

    // Each thread takes one ray.

    // Combined load of the data for the snode to shared memory.

    // Test all 16 child node BBoxes against every ray.
 
    // If hit:
    // (TODO) Add to the queue for that outlet.

    // If miss:
    // Leave it. 


    int sampleCount = cuConstRendererParams.sampleCount;
    int imageWidth = cuConstRendererParams.imageWidth;
    int imageHeight = cuConstRendererParams.imageHeight;
    int rayCount = imageWidth * imageHeight * sampleCount;

    if(index < rayCount) {
        return;
    }

    CuRay *r = &cuConstRendererParams.queues[rayCount * snode + index];
    CuRay *raylist = &cuConstRendererParams.queues[rayCount * snode];
    
    bool is_leaf = (cuConstRendererParams.bvhSubTrees[snode].outlets[index] == TREE_WIDTH);
    // Combined load.
    if(!is_leaf) {
        __shared__ CuBVHSubTree subtree;
        __shared__ uint _outlets[TREE_WIDTH * RAYS_PER_BLOCK];
        __shared__  uint _c_outlets[TREE_WIDTH * RAYS_PER_BLOCK];
        __shared__ uint _compacter[TREE_WIDTH * RAYS_PER_BLOCK];
        __shared__ uint _c_qid[TREE_WIDTH * RAYS_PER_BLOCK];
        //__shared__ uint _scratch[TREE_WIDTH * RAYS_PER_BLOCK];
        
        if(index < 1) {
            subtree.start = cuConstRendererParams.bvhSubTrees[snode].start;
            subtree.range = cuConstRendererParams.bvhSubTrees[snode].range;
        }

        if(index < TREE_WIDTH) {
            subtree.outlets[index] = cuConstRendererParams.bvhSubTrees[snode].outlets[index];

            subtree.minl[index * 2 + 0] = cuConstRendererParams.bvhSubTrees[snode].minl[index * 2 + 0];
            subtree.minl[index * 2 + 1] = cuConstRendererParams.bvhSubTrees[snode].minl[index * 2 + 1];
            subtree.maxl[index * 2 + 0] = cuConstRendererParams.bvhSubTrees[snode].maxl[index * 2 + 0];
            subtree.maxl[index * 2 + 1] = cuConstRendererParams.bvhSubTrees[snode].maxl[index * 2 + 1];
        }

        __syncthreads();


        for(int i = 0; i < TREE_WIDTH; i++) {
            // Intersect the rays here.
            float t = intersectBBox(r->o, r->d, subtree.minl[i], subtree.maxl[i]);

            // TODO: Make sure we account for points inside the box too.

            if( t >= 0 ) {
                // If intersected, place a mark.
                _outlets[i * RAYS_PER_BLOCK + index] = 1;
            }
        }

        // Perform compaction on every 512x:(512x + 512)
        //_assignments[threadIdx.x] = assignments[(blockIdx.x << sizelog) + threadIdx.x];
        //if(blockIdx.x == 1000 && _assignments[threadIdx.x]){// && _assignments[(blockIdx.x << sizelog) + threadIdx.x]) {
        //    printf("%d: %u\n", threadIdx.x, blockIdx.x << sizelog);
        //}
        //_compacter[threadIdx.x] = compacter[blockIdx.x * size + threadIdx.x];
        //if(threadIdx.x % 10 == 0)
        //    printf("hello\n");
        // Wait for sync.
        __syncthreads();


        //sharedMemExclusiveScan(thread, _assignments, _compacter[sliceIdx * size], &scratch[sliceIdx * size], size);
        for(int i = 0; i < TREE_WIDTH; i++) {
            sharedMemExclusiveScan(threadIdx.x, &_outlets[i * RAYS_PER_BLOCK], &_c_outlets[i * RAYS_PER_BLOCK], &_c_qid[i * RAYS_PER_BLOCK], RAYS_PER_BLOCK);
        }

        __syncthreads();

        // Rearrange.
        for(int i = 0; i < TREE_WIDTH; i++) {
            if(index >= RAYS_PER_BLOCK) 
                continue;

            uint k0 = _c_outlets[i * RAYS_PER_BLOCK + index + 0];
            uint k1 = _c_outlets[i * RAYS_PER_BLOCK + index + 1];
            

            if(index != RAYS_PER_BLOCK - 1) {
                if(k0 + 1 == k1) 
                    _c_qid[k0] = index;
            } else {
                if(_outlets[i * RAYS_PER_BLOCK + index])
                    _c_qid[k0] = index;
            }
        }

        // Write out.
        for(int i = 0; i < TREE_WIDTH; i++) {
            int target = subtree.outlets[i];

            int rayid = _c_qid[i * RAYS_PER_BLOCK + index];

            if(index < _c_outlets[(i+1) * RAYS_PER_BLOCK - 1]) {
                // Leave.
                break;
            }

            __shared__ int tindex;

            __syncthreads();
            // Atomic grab.
            if (i == 0) {
                tindex = atomicAdd(&cuConstRendererParams.qCounts[target], _c_outlets[(i+1) * RAYS_PER_BLOCK - 1]);
            }

            __syncthreads();


            cuConstRendererParams.queues[rayCount * target + tindex + index] = raylist[rayid];
        }

    } else {
        // This is a leaf node. Quickly load whatever all triangles.
        __shared__ CuTriangle _triangles[MAX_TRIANGLES];
        int num_triangles = cuConstRendererParams.bvhSubTrees[snode].range;

        // Copy to shared memory
        if(index < num_triangles) {
            _triangles[index] = cuConstRendererParams.triangles[cuConstRendererParams.bvhSubTrees[snode].start + index];
        }

        float t = MAX_T_DISTANCE;

        CuTriangle tri;
        // Perform triangle intersect.
        for(int i = 0; i < num_triangles; i++) {
            float thist;
            if((thist = intersectRayTriangle(_triangles[i].a, _triangles[i].b, _triangles[i].c, r->o, r->d)) < t){
                t = thist;
                tri = _triangles[i];
            }

        }

        int x = static_cast<int>(r->ss.x);
        int y = static_cast<int>(r->ss.y);

        int sid = r->sid;

        int imageWidth = cuConstRendererParams.imageWidth;
        int imageHeight = cuConstRendererParams.imageHeight;
        
        //float4 *fx = &imageData[((y * height + x) * sampleCount + sid)];
        CuIntersection *its = &cuConstRendererParams.intersections[((y * imageHeight+ x) * sampleCount + sid)];
        if(its->t < t) {
            return;
        }

        bool direct_light = !(r->lightImportance == make_float3(0.0, 0.0, 0.0));
        if(!direct_light) {
            // Overwrite the intersection.
            its->t = t;
            its->pt = r->o + r->d * t;
            //its->lightImportance = r->lightImportance;
            its->light = r->light;
            its->importance = r->importance;

            //float3 n = normalize(cross(tri.a - tri.b, tri.b - tri.c));
            //its->n = ((dot(n, r->d) < 0) ? -1 : 1) * n;

            // Compute barycentric coordinates.
            float total = length(cross(tri.a - tri.b, tri.b - tri.c));
            //float bC = 0.0;
            //float bA = 0.0;
            //float bB = 0.0;

            float bC = length(cross(tri.a - its->pt, tri.b - its->pt)) / total;
            float bA = length(cross(tri.b - its->pt, tri.c - its->pt)) / total;
            float bB = length(cross(tri.c - its->pt, tri.a - its->pt)) / total;
            its->n = bA * tri.n0 + bB * tri.n1 + bC * tri.n2;

            // Make 2 more axes.
            float3 ax = normalize(cross(make_float3(0.1, 0.1, 1), its->n));
            float3 ay = normalize(cross(ax, its->n));
            its->wi = normalize(make_float3(dot(ax, r->d), dot(ay, r->d), dot(its->n, r->d)));

            its->ss = r->ss;
            its->sid = r->sid;
            its->bsdf = tri.bsdf;
        } else {
            // If direct light estimate, then only estimate the light at this point.
            its->light = r->light + ((t < r->maxT) ? (r->lightImportance) : make_float3(0.0)); // TODO: Make update.

        }

    }


}

__global__ void kernelRayIntersectSingle(int snode) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    rayIntersectSingle(snode, index);
}

// Intersection function.
// Performs ray intersection on a full level.
__global__ void kernelRayIntersectLevel(int level) {
    // Mapping: Each block takes a subset of rays split for that level.
    // Use blockID bits to split the rays to each snode in the level.
    // Each block only works on one particular snode.

    // Each thread takes one ray.

    // Combined load of the data for the snode to shared memory.

    // Test all 16 child node BBoxes against every ray.

    // If hit:
    // (TODO) Add to the queue for that outlet.

    // If miss:
    // Leave it.
    
    // Use a table to compute the nodes at this level.
    
    // Compute the queue index.
    int levelIndex = (blockIdx.x * blockDim.x) >> QUEUE_LENGTH_LOG2;
    
    int nodeIndex = cuConstRendererParams.levelIndices[level * LEVEL_INDEX_SIZE + levelIndex];
    int rayIndex = (blockIdx.x * blockDim.x) & (QUEUE_LENGTH_LOG2 - 1);

    rayIntersectSingle(nodeIndex, rayIndex);
}


    //#define NUM_CIRCLES_PER_BLOCK 1024
#define NUM_LAYERS 3

    //#define LOG_CIRCLES_PER_BLOCK 13
    CudaRenderer::CudaRenderer() {
        image = NULL;
        deviceBSDFs = NULL;
        deviceEmitters = NULL;
        deviceBVHSubTrees = NULL;
        deviceTriangles = NULL;
        deviceRays = NULL;
        deviceIntersections = NULL;
        deviceImageData = NULL;
        deviceSSImageData = NULL;
        deviceLevelIndices = NULL;
    }

    CudaRenderer::~CudaRenderer() {

        if (image) {
            delete image;
        }

        if (bsdfs.size() > 0) {
            bsdfs.clear();
            emitters.clear();
            subtrees.clear();
            triangles.clear();
            delete [] levelIndices;
        }

        if (deviceBSDFs) {
            hipFree(deviceBSDFs);
            hipFree(deviceEmitters);
            hipFree(deviceBVHSubTrees);
            hipFree(deviceTriangles);   
            hipFree(deviceRays);
            hipFree(deviceIntersections);
            hipFree(deviceSSImageData);
            hipFree(deviceImageData);
            hipFree(deviceLevelIndices);
        }
    }

    const Image* CudaRenderer::getImage() {

        // Need to copy contents of the rendered image from device memory
        // before we expose the Image object to the caller

        printf("Copying image data from device\n");

        hipMemcpy(image->data,
                deviceImageData,
                sizeof(float) * 4 * image->width * image->height,
                hipMemcpyDeviceToHost);

        return image;
    }
    
    DynamicScene::Scene* CudaRenderer::loadFromSceneInfo(Collada::SceneInfo* sceneInfo) {
    
  vector<Collada::Node> &nodes = sceneInfo->nodes;
  vector<DynamicScene::SceneLight *> lights;
  vector<DynamicScene::SceneObject *> objects;

  // save camera position to update camera control later
  Collada::CameraInfo *c;
  Vector3D c_pos = Vector3D();
  Vector3D c_dir = Vector3D();

  int len = nodes.size();
  for (int i = 0; i < len; i++) {
    Collada::Node &node = nodes[i];
    Collada::Instance *instance = node.instance;
    const Matrix4x4 &transform = node.transform;

    switch (instance->type) {
      case Collada::Instance::CAMERA:
        c = static_cast<Collada::CameraInfo *>(instance);
        c_pos = (transform * Vector4D(c_pos, 1)).to3D();
        c_dir = (transform * Vector4D(c->view_dir, 1)).to3D().unit();
        init_camera(*c, transform);
        break;
      case Collada::Instance::LIGHT: {
        lights.push_back(
            init_light(static_cast<Collada::LightInfo &>(*instance), transform));
        break;
      }
      case Collada::Instance::SPHERE:
        objects.push_back(
            init_sphere(static_cast<Collada::SphereInfo &>(*instance), transform));
        break;
      case Collada::Instance::POLYMESH:
        objects.push_back(
            init_polymesh(static_cast<Collada::PolymeshInfo &>(*instance), transform));
        break;
      case Collada::Instance::MATERIAL:
        //init_material(static_cast<Collada::MaterialInfo &>(*instance));
        std::cout << "Unable to handle material.\n" << std::endl;
        break;
    }
  }

	// TODO: TEmporarily disabled this to test environment lights.
  if (lights.size() == 0) {  // no lights, default use ambient_light
    Collada::LightInfo default_light = Collada::LightInfo();
    lights.push_back(new DynamicScene::AmbientLight(default_light));
  }
  DynamicScene::Scene* scene = new DynamicScene::Scene(objects, lights);

  const BBox &bbox = scene->get_bbox();
  if (!bbox.empty()) {
    //Vector3D target = bbox.centroid();
    //canonical_view_distance = bbox.extent.norm() / 2 * 1.5;

    //double view_distance = canonical_view_distance * 2;
    //double min_view_distance = canonical_view_distance / 10.0;
    //double max_view_distance = canonical_view_distance * 20.0;

    //canonicalCamera.place(target, acos(c_dir.y), atan2(c_dir.x, c_dir.z),
    //                      view_distance, min_view_distance, max_view_distance);

    //camera.place(target, acos(c_dir.y), atan2(c_dir.x, c_dir.z), view_distance,
    //             min_view_distance, max_view_distance);

    //set_scroll_rate();
  }

  // set default draw styles for meshEdit -
  //scene->set_draw_styles(&defaultStyle, &hoverStyle, &selectStyle);

  // cerr << "==================================" << endl;
  // cerr << "CAMERA" << endl;
  // cerr << "      hFov: " << camera.hFov << endl;
  // cerr << "      vFov: " << camera.vFov << endl;
  // cerr << "        ar: " << camera.ar << endl;
  // cerr << "     nClip: " << camera.nClip << endl;
  // cerr << "     fClip: " << camera.fClip << endl;
  // cerr << "       pos: " << camera.pos << endl;
  // cerr << " targetPos: " << camera.targetPos << endl;
  // cerr << "       phi: " << camera.phi << endl;
  // cerr << "     theta: " << camera.theta << endl;
  // cerr << "         r: " << camera.r << endl;
  // cerr << "      minR: " << camera.minR << endl;
  // cerr << "      maxR: " << camera.maxR << endl;
  // cerr << "       c2w: " << camera.c2w << endl;
  // cerr << "   screenW: " << camera.screenW << endl;
  // cerr << "   screenH: " << camera.screenH << endl;
  // cerr << "screenDist: " << camera.screenDist<< endl;
  // cerr << "==================================" << endl;
        return scene;
    }

    void CudaRenderer::loadScene(std::string sceneFilePath) {
        //sceneName = scene;
        
        Collada::SceneInfo* sceneInfo = new Collada::SceneInfo();
        if (Collada::ColladaParser::load(sceneFilePath.c_str(), sceneInfo) < 0) {
            printf("Error: parsing failed!\n");
            delete sceneInfo;
            exit(0);
        }
    
        DynamicScene::Scene* dscene = this->loadFromSceneInfo(sceneInfo);
        StaticScene::Scene* scene = dscene->get_static_scene();

        std::vector<StaticScene::Primitive *> primitives;
        for (StaticScene::SceneObject *obj : scene->objects) {
            const vector<StaticScene::Primitive *> &obj_prims = obj->get_primitives();
            primitives.reserve(primitives.size() + obj_prims.size());
            primitives.insert(primitives.end(), obj_prims.begin(), obj_prims.end());
        }
        
        //std::vector<CuTriangle> cuts;
        for(auto prim : primitives) {
           StaticScene::Triangle* t = reinterpret_cast<StaticScene::Triangle*>(prim); 
           Vector3D v0,v1,v2;
           t->positions(v0, v1, v2);
           Vector3D n0,n1,n2;
           t->normals(n0, n1, n2);

           CuTriangle ct;
           ct.a = make_float3(v0.x, v0.y, v0.z);
           ct.b = make_float3(v1.x, v1.y, v1.z);
           ct.c = make_float3(v2.x, v2.y, v2.z);

           ct.n0 = make_float3(n0.x, n0.y, n0.z);
           ct.n1 = make_float3(n1.x, n1.y, n1.z);
           ct.n2 = make_float3(n2.x, n2.y, n2.z);

           ct.bsdf = 0;
           ct.emit = -1; 

           triangles.push_back(ct);
        }
        
        // Add BSDFs
        // TODO: Make this automated soon.
        //std::vector<CuBSDF> bsdfs;
        CuBSDF b;
        b.albedo = make_float3(0.6, 0.6, 0.6);
        b.fn = 0;
        b.nu = 0;

        bsdfs.push_back(b);

        // Add Emitters
        // TODO: Make this automated soon.
        CuEmitter e;
        auto l = scene->lights[0];
        auto al = reinterpret_cast<StaticScene::AreaLight*>(l);
        e.position = v2f3(al->position);
        e.direction = v2f3(al->direction);
        e.dim_x = v2f3(al->dim_x);
        e.dim_y = v2f3(al->dim_y);
        e.radiance = make_float3(al->radiance.r, al->radiance.g, al->radiance.b);

        //std::vector<CuEmitter> emitters;
        emitters.push_back(e);


        auto bvh = new StaticScene::BVHAccel(primitives);
        
        int scount;
        //this->subtrees = bvh->compressedTree();
        auto tmp_subtrees = bvh->compactedTree(); // Tree compaction system to make the tree smaller.

        std::vector<StaticScene::C_BVHSubTree> tree;
        std::vector<int> levelCounts;
        
        this->levelIndices = (int*) malloc(sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS);
        tmp_subtrees->compress(tree, this->levelIndices, LEVEL_INDEX_SIZE, levelCounts, 0, MAX_LEVELS); // Compressed subtree.

        for(auto entry : tree) {
            CuBVHSubTree cutree;
            cutree.start = entry.start;
            cutree.range = entry.range;
            for(int i = 0; i < TREE_WIDTH; i++){
                cutree.minl[i] = v2f3(entry.min[i]);
                cutree.maxl[i] = v2f3(entry.max[i]);
                cutree.outlets[i] = entry.outlets[i];
            }
            this->subtrees.push_back(cutree);
        }
        
    }
    
    void CudaRenderer::setup() {

        int deviceCount = 0;
        bool isFastGPU = false;
        std::string name;
        hipError_t err = hipGetDeviceCount(&deviceCount);

        printf("---------------------------------------------------------\n");
        printf("Initializing CUDA for CudaRenderer\n");
        printf("Found %d CUDA devices\n", deviceCount);

        for (int i=0; i<deviceCount; i++) {
            hipDeviceProp_t deviceProps;
            hipGetDeviceProperties(&deviceProps, i);
            name = deviceProps.name;
            if (name.compare("GeForce GTX 1040") == 0)
            {
                isFastGPU = true;
            }

            printf("Device %d: %s\n", i, deviceProps.name);
            printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
            printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
            printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        }
        printf("---------------------------------------------------------\n");
        if (!isFastGPU)
        {
            printf("WARNING: "
                    "You're not running on a fast GPU, please consider using "
                    "NVIDIA GTX 480, 670 or 780.\n");
            printf("---------------------------------------------------------\n");
        }

        // By this time the scene should be loaded.  Now copy all the key
        // data structures into device memory so they are accessible to
        // CUDA kernels
        //
        // See the CUDA Programmer's Guide for descriptions of
        // hipMalloc and hipMemcpy
        // Compute Pow2 and Log2 versions of numCircles, this is important for the compaction algorithm,
        // which only works with powers of 2/


        // TODO: WARN: Temporary override.
        //numCircles = 1024;
        
        int numRays = SAMPLES_PER_PIXEL * image->width * image->height;
        int queueSize = numRays * subtrees.size();

        hipMalloc(&deviceBSDFs, sizeof(CuBSDF) * bsdfs.size());
        hipMalloc(&deviceEmitters, sizeof(CuEmitter) * emitters.size());
        hipMalloc(&deviceTriangles, sizeof(CuTriangle) * triangles.size());
        hipMalloc(&deviceBVHSubTrees, sizeof(CuBVHSubTree) * subtrees.size());
        hipMalloc(&deviceRays, sizeof(CuRay) * queueSize);
        hipMalloc(&deviceIntersections, sizeof(CuIntersection) * numRays);
        hipMalloc(&deviceLevelIndices, sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS); 
        hipMalloc(&deviceSSImageData, sizeof(float) * 4 * image->width * image->height * SAMPLES_PER_PIXEL);
        hipMalloc(&deviceImageData, sizeof(float) * 4 * image->width * image->height);

        //int* hcounts = reinterpret_cast<int*>(calloc((image->width >> KWIDTH) * (image->height >> KWIDTH), sizeof(int)));

        //hipMemcpy(counts, hcounts, sizeof(int) * (image->width >> KWIDTH) * (image->height >> KWIDTH), hipMemcpyHostToDevice);
        //hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
        //hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
        //hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
        //hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);
        hipMemcpy(deviceBSDFs, &bsdfs[0], sizeof(CuBSDF) * bsdfs.size(), hipMemcpyHostToDevice);
        hipMemcpy(deviceEmitters, &emitters[0], sizeof(CuEmitter) * emitters.size(), hipMemcpyHostToDevice);
        hipMemcpy(deviceTriangles, &triangles[0], sizeof(CuTriangle) * triangles.size(), hipMemcpyHostToDevice);
        hipMemcpy(deviceBVHSubTrees, &subtrees[0], sizeof(CuBVHSubTree) * subtrees.size(), hipMemcpyHostToDevice);


        // Initialize parameters in constant memory.  We didn't talk about
        // constant memory in class, but the use of read-only constant
        // memory here is an optimization over just sticking these values
        // in device global memory.  NVIDIA GPUs have a few special tricks
        // for optimizing access to constant memory.  Using global memory
        // here would have worked just as well.  See the Programmer's
        // Guide for more information about constant memory.

        GlobalConstants params;

        // Compute Pow2 and Log2 versions of numCircles, this is important for the compaction algorithm,
        // which only works with powers of 2/
        //int powlevel = 0;
        //int temp = numCircles;
        //while(temp >>= 1) powlevel ++;
        //params.numCirclesLog2 = powlevel+1;
        //params.numCirclesPow2 = 1 << (powlevel+1);
        //printf("params.numCirclesPow2 %d\n", 1 << (powlevel + 1));
        params.imageWidth = image->width;
        params.imageHeight = image->height;
        params.bsdfs = deviceBSDFs;
        params.emitters = deviceEmitters;
        params.bvhSubTrees = deviceBVHSubTrees;
        params.triangles = deviceTriangles;
        params.queues = deviceRays;
        params.intersections = deviceIntersections;
        params.ssImageData = (float4*)deviceSSImageData;
        params.imageData = (float4*)deviceImageData;

        hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

        // Also need to copy over the noise lookup tables, so we can
        // implement noise on the GPU
        int* permX;
        int* permY;
        float* value1D;
        //getNoiseTables(&permX, &permY, &value1D);
        //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
        //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
        //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

        // Copy over the color table that's used by the shading
        // function for circles in the snowflake demo

        /*float lookupTable[COLOR_MAP_SIZE][3] = {
            {1.f, 1.f, 1.f},
            {1.f, 1.f, 1.f},
            {.8f, .9f, 1.f},
            {.8f, .9f, 1.f},
            {.8f, 0.8f, 1.f},
        };

        hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);*/
        printf("Finished allocation and copy\n");
    }

    // allocOutputImage --
    //
    // Allocate buffer the renderer will render into.  Check status of
    // image first to avoid memory leak.
    void CudaRenderer::allocOutputImage(int width, int height) {
        printf("Image alloc\n");
        if (image)
            delete image;
        image = new Image(width, height);
        printf("Done Image alloc\n");
    }

    // clearImage --
    //
    // Clear the renderer's target image.  The state of the image after
    // the clear depends on the scene being rendered.
    void CudaRenderer::clearImage() {

        dim3 blockDim(16, 16, 1);
        dim3 gridDim(
                (image->width + blockDim.x - 1) / blockDim.x,
                (image->height + blockDim.y - 1) / blockDim.y);

        kernelClearImage<<<gridDim, blockDim>>>(0.f, 0.f, 0.f, 0.f);

        hipDeviceSynchronize();
        printf("Done cleaning\n");
    }

    void CudaRenderer::render() {
        //printf("Started rendering %d\n", batchSize);fflush(stdout);
        // 256 threads per block is a healthy number
        //dim3 blockDim(NUM_THREADS, 1);
        //dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);
        //dim3 gridDim(numCircles);

        //dim3 blockDim(1024);
        //dim3 gridDim(1024 * (NUM_CIRCLES_PER_BLOCK >> 10));
        //dim3 blockDim((batchSize > 1024) ? 1024 : batchSize);
        //dim3 gridDim(1024 * ((numCircles > 1024) ? numCircles >> 10 : 1));


        //double start = CycleTimer::currentSeconds();

        //kernelClearBuffers<<<gridDimCl, blockDimCl>>>((float3*)qPositions, qRadii, (float3*)qColors, queues);
        //hipDeviceSynchronize();

        //double clear = CycleTimer::currentSeconds();
        //printf("Executing kernel. %d, %d\n", numCircles, NUM_THREADS);
/*        kernelRenderCircles<<<gridDim, blockDim>>>(assignments, counts);
        hipDeviceSynchronize(); 

        //double render = CycleTimer::currentSeconds();

        kernelExScan<<<gridDim3, blockDim3>>>((uint*)(assignments), (uint*)(compactor), (uint*)(scratch));
        hipDeviceSynchronize();

        //double exscan = CycleTimer::currentSeconds();

        kernelCompact<<<gridDimC, blockDimC>>>((uint*)(assignments), (uint*)(compactor), (uint*)(queues), (float3*)qPositions, (float*)qRadii, (float3*)qColors, (int*) layerCount);
        hipDeviceSynchronize();

        //double compact = CycleTimer::currentSeconds();

        if(sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME){
            //printf("Snowflake rendering.\n");
            kernelRenderSnowPixels<<<gridDim2, blockDim2>>>((uint*)queues, (float3*)qPositions, (float*)qRadii, (float3*)qColors, (float4*)layered, (int*) layerCount);
        }else{
            kernelRenderPixels<<<gridDim2, blockDim2>>>((uint*)queues, (float3*)qPositions, (float*)qRadii, (float3*)qColors, (float4*)layered, (int*) layerCount); 
        }
        hipDeviceSynchronize();*/

        //double pixrender = CycleTimer::currentSeconds();


        //double combine = CycleTimer::currentSeconds();

        /*printf("ClearBuffers: %.4fms \n", (clear - start) * 1000.f);
          printf("RenderCircles: %.4fms \n", (render - clear) * 1000.f);
          printf("ExScan: %.4fms \n", (exscan - render) * 1000.f);
          printf("Compact: %.4fms \n", (compact - exscan) * 1000.f);
          printf("RenderPixels: %.4fms \n", (pixrender - compact) * 1000.f);
          printf("Combine: %.4fms \n", (combine - pixrender) * 1000.f);
          printf("Total Render: %.4fms \n", (combine - start) * 1000.f);*/

        //short* host_a = reinterpret_cast<short*>(malloc(sizeof(short) * 32 * 32 * numCircles));
        //int* host_c = reinterpret_cast<int*>(malloc(sizeof(int) *  32 * 32 ));
        //hipMemcpy(host_a, assignments, sizeof(short) * 32 * 32 * numCircles, hipMemcpyDeviceToHost);
        //hipMemcpy(host_c, counts, sizeof(int) * 32 * 32, hipMemcpyDeviceToHost);
        //printf("Item at : %d\n",host_c[5]);
        //printf("Finished rendering\n");
        //printf("Done rendering\n");fflush(stdout);
        //hipDeviceSynchronize();
    }

}
