#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include "cudaRenderer.h"
#include "cuda_image.h"
//#include "noise.h"
//#include "sceneLoader.h"
//#include "util.h"
//#include "cycleTimer.h"

#define SCAN_BLOCK_DIM 512  // needed by sharedMemExclusiveScan implementation
#include "exclusiveScan.cu_inl"
#include "samplers.cu_inl"
#include "cuda_util.h"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

namespace cutracer {

    float3 v2f3(Vector3D v) {
        return make_float3(v.x, v.y, v.z);
    }

    Vector3D f32v(float3 f) {
        return Vector3D(f.x, f.y, f.z);
    }
    void init_camera(Collada::CameraInfo &cameraInfo,
            const Matrix4x4 &transform) {
        //camera.configure(cameraInfo, screenW, screenH);
        //canonicalCamera.configure(cameraInfo, screenW, screenH);
        //set_projection_matrix();
    }

    DynamicScene::SceneLight *init_light(Collada::LightInfo &light,
            const Matrix4x4 &transform) {
        switch (light.light_type) {
            case Collada::LightType::NONE:
                break;
            case Collada::LightType::AMBIENT:
                return new DynamicScene::AmbientLight(light);
            case Collada::LightType::DIRECTIONAL:
                return new DynamicScene::DirectionalLight(light, transform);
            case Collada::LightType::AREA:
                return new DynamicScene::AreaLight(light, transform);
            case Collada::LightType::POINT:
                return new DynamicScene::PointLight(light, transform);
            case Collada::LightType::SPOT:
                return new DynamicScene::SpotLight(light, transform);
            default:
                break;
        }
        return nullptr;
    }

    /**
     * The transform is assumed to be composed of translation, rotation, and
     * scaling, where the scaling is uniform across the three dimensions; these
     * assumptions are necessary to ensure the sphere is still spherical. Rotation
     * is ignored since it's a sphere, translation is determined by transforming the
     * origin, and scaling is determined by transforming an arbitrary unit vector.
     */
    DynamicScene::SceneObject *init_sphere(
            Collada::SphereInfo &sphere, const Matrix4x4 &transform) {
        const Vector3D &position = (transform * Vector4D(0, 0, 0, 1)).projectTo3D();
        double scale = (transform * Vector4D(1, 0, 0, 0)).to3D().norm();
        return new DynamicScene::Sphere(sphere, position, scale);
    }

    DynamicScene::SceneObject *init_polymesh(
            Collada::PolymeshInfo &polymesh, const Matrix4x4 &transform) {
        return new DynamicScene::Mesh(polymesh, transform);
    }

    struct GlobalConstants {

        // Image data.
        int imageWidth;
        int imageHeight;
        float4* ssImageData; // Super sampled image data.
        float4* imageData; // Final image data.
        int sampleCount;

        // Camera data.
        float3 c_origin;
        float3 c_lookAt;
        float3 c_up;
        float3 c_left;

        // Ray queues: SxR (S=Number of subtrees, R=Max number of rays per queue) 
        // (Gigantic ~32M entries)
        // Two buffers for double buffering
        CuRay* queues1;
        CuRay* queues2;

        // Queue counts. Initialized to 0.
        uint* qCounts;

        // Ray intersection buffers.
        // (Gigantic ~32M entries)
        CuIntersection* intersections;

        // Triangle list. (Huge 1000-1000000 entries)
        CuTriangle* triangles;

        // Emitter list. (Tiny 1-2 entries)
        CuEmitter* emitters;

        // BSDF list. (Small ~10 entries).
        CuBSDF* bsdfs;

        // BVHNode list
        // (Large 10-4000 entries)
        CuBVHSubTree* bvhSubTrees;

        int* levelIndices;

        // Intersection atomicity helpers.
        CuIntersection* multiIntersections;

        uint* intersectionTokens;

        float* minT;
        
        hiprandState* randomStates;
    };

    // Global variable that is in scope, but read-only, for all cuda
    // kernels.  The __constant__ modifier designates this variable will
    // be stored in special "constant" memory on the GPU. (we didn't talk
    // about this type of memory in class, but constant memory is a fast
    // place to put read-only variables).
    __constant__ GlobalConstants cuConstRendererParams;

    __device__ __inline__ void cudaswap(float& a, float& b) {
        float temp = a;
        a = b;
        b = temp;
    }

    __device__ __inline__ float intersectBBox(float3 o, float3 d, float3 min, float3 max) {
        
        if((o.x >= min.x && o.x <= max.x) 
            && (o.y >= min.y && o.y <= max.y) 
            && (o.z >= min.z && o.z <= max.z))
            return 0.0;

        float tmin = (min.x - o.x) / d.x; 
        float tmax = (max.x - o.x) / d.x; 

        //printf("tmin: %f\n", tmin);
        //printf("tmax: %f\n\n", tmax);
        if (tmin > tmax) cudaswap(tmin, tmax); 

        float tymin = (min.y - o.y) / d.y; 
        float tymax = (max.y - o.y) / d.y; 
        //printf("tymin: %f\n", tymin);
        //printf("tymax: %f\n\n", tymax);

        if (tymin > tymax) cudaswap(tymin, tymax); 

        if ((tmin > tymax) || (tymin > tmax)) 
            return -1.0; 

        if (tymin > tmin) 
            tmin = tymin; 
        //printf("tmin: %f\n", tmin);
        //printf("tmax: %f\n\n", tmax);

        if (tymax < tmax) 
            tmax = tymax; 

        float tzmin = (min.z - o.z) / d.z; 
        float tzmax = (max.z - o.z) / d.z; 
        //printf("tzmin: %f\n", tzmin);
        //printf("tzmax: %f\n\n", tzmax);

        if (tzmin > tzmax) cudaswap(tzmin, tzmax); 

        if ((tmin > tzmax) || (tzmin > tmax)) 
            return -1.0;

        if (tzmin > tmin) 
            tmin = tzmin; 

        //printf("tmin: %f\n", tmin);
        //printf("tmax: %f\n", tmax);

        if (tzmax < tmax) 
            tmax = tzmax; 

        return tmin;

    }

    /*__device__ __inline__ float3 crossProduct(float3 a, float3 b) {
      return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x - b.z, a.x * b.y - a.y * b.x);
      }

      __device__ __inline__ float dotProduct(float3 a, float3 b) {
      return a.x * b.x + a.y * b.y + a.z * b.z;
      }*/

    __device__ __inline__ float intersectRayTriangle(float3 v0, float3 v1, float3 v2, float3 orig, float3 dir) {
        //float e1[3],e2[3],h[3],s[3],q[3];
        //float a,f,u,v;
        //vector(e1,v1,v0);
        //vector(e2,v2,v0);
        // compute plane's normal
        float3 v0v1 = v1 - v0; 
        float3 v0v2 = v2 - v0; 
        // no need to normalize
        float3 N = cross(v0v1, v0v2); // N 
        float area2 = length(N);

        // Step 1: finding P

        // check if ray and plane are parallel ?
        float NdotRayDirection = dot(N, dir); 
        if (abs(NdotRayDirection) < 1e-6) // almost 0 
            return -1; // they are parallel so they don't intersect ! 

        // compute d parameter using equation 2
        float d = dot(N, v0); 

        // compute t (equation 3)
        float t = (d - dot(N, orig)) / NdotRayDirection; 

        // check if the triangle is in behind the ray
        if (t < 0) return t; // the triangle is behind 

        // compute the intersection point using equation 1
        float3 P = orig + t * dir; 

        // Step 2: inside-outside test
        float3 C; // vector perpendicular to triangle's plane 

        // edge 0
        float3 edge0 = v1 - v0;
        float3 vp0 = P - v0;
        C = cross(edge0, vp0);
        if (dot(N, C) < 0) return -3; // P is on the right side 

        // edge 1
        float3 edge1 = v2 - v1; 
        float3 vp1 = P - v1; 
        C = cross(edge1, vp1); 
        if (dot(N, C) < 0)  return -4; // P is on the right side 

        // edge 2
        float3 edge2 = v0 - v2; 
        float3 vp2 = P - v2; 
        C = cross(edge2, vp2); 
        if (dot(N, C) < 0) return -5; // P is on the right side; 

        return t; // this ray hits the triangle 
    }

    __global__ void kernelClearBuffers(float3* positions, float* radii, float3* colors, int* queues) {
        //int idx = threadIdx.x;
        //int block = blockIdx.x;
        //int layer = blockIdx.y;

        //positions[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = make_float3(0.f,0.f,0.f);
        //radii[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = 0.f;
        //colors[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = make_float3(0.f,0.f,0.f);
        //queues[idx + blockDim.x * block + blockDim.x * gridDim.x * layer] = 0;
    }
    // kernelClearImage --  (CUDA device code)
    //
    // Clear the image, setting all pixels to the specified color rgba
    __global__ void kernelClearImage(float r, float g, float b, float a) {
        int imageX = blockIdx.x * blockDim.x + threadIdx.x;
        int imageY = blockIdx.y * blockDim.y + threadIdx.y;

        int width = cuConstRendererParams.imageWidth;
        int height = cuConstRendererParams.imageHeight;

        if (imageX >= width || imageY >= height)
            return;

        int offset = (imageY * width + imageX);
        float4 value = make_float4(r, g, b, a);

        // Write to global memory: As an optimization, this code uses a float4
        // store, which results in more efficient code than if it were coded as
        // four separate float stores.
        *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
    }



    // PT functions.

    // Generate camera rays and insert into queue.
    __global__ void kernelPrimaryRays( ) {

        // For each pixel
        // For each sample per pixel
        // Create ray and set to queue at the falling icicomputed offset.

        int imageX = blockIdx.x * blockDim.x + threadIdx.x;
        int imageY = blockIdx.y * blockDim.y + threadIdx.y;

        int width = cuConstRendererParams.imageWidth;
        int height = cuConstRendererParams.imageHeight;

        //int raycount = cuConstRendererParams.rayCount;
        //printf("x:%d, y:%d\n", imageX, imageY);
        
        int id = imageX * height + imageY;

        int sampleCount = cuConstRendererParams.sampleCount;
        hiprandState rand = cuConstRendererParams.randomStates[0];

        for(int i = 0; i < sampleCount; i++) {
            int destIndex = i + (imageX * height + imageY) * sampleCount;

            //float2 sample = make_float2(0.5f, 0.5f);
            float2 sample = squareSample(&rand);

            float xs = imageX + sample.x;
            float ys = imageY + sample.y;

            float2 ss = make_float2(xs, ys);

            //float a = (ss.x / width) - 0.5;
            //float b = (ss.y / height) - 0.5;
            //float c = sqrt(1 - (a*a + b*b));
            
            float3 k = make_float3((ss.y/width)-0.5, -((ss.x/height)-0.5),1.0);
            k = k / length(k);

            //float px = a * cuConstRendererParams.c_left.x + b * cuConstRendererParams.c_up.x + c * cuConstRendererParams.c_lookAt.x;
            //float py = a * cuConstRendererParams.c_left.y + b * cuConstRendererParams.c_up.y + c * cuConstRendererParams.c_lookAt.y;
            //float pz = a * cuConstRendererParams.c_left.z + b * cuConstRendererParams.c_up.z + c * cuConstRendererParams.c_lookAt.z;

            float3 dir = k.x * cuConstRendererParams.c_left + k.y * cuConstRendererParams.c_up + k.z * cuConstRendererParams.c_lookAt;
            //float3 dir = 
            //printf("x:%d, y:%d : %f %f %f\n", imageX, imageY, px, py, pz);

            CuRay *r = &cuConstRendererParams.queues1[destIndex];
            r->o = cuConstRendererParams.c_origin;
            r->d = dir;
            r->importance = make_float3(1, 1, 1);
            r->lightImportance = make_float3(0, 0, 0);
            r->light = make_float3(0, 0, 0);
            r->maxT = INFINITY; // TODO: Put inf.
            r->ss = ss;
            r->sid = i;
            r->id = destIndex;
            r->valid = true;
        }

        cuConstRendererParams.randomStates[0] = rand;

    }

    // Generate direct light rays from intersections.
    __global__ void kernelDirectLightRays() {
        // For each element in intersection. (Map each intersection to a thread).
        // For each light
        // Create a ray from light to intersection.
        // Add ray to queue.

        int iid = blockIdx.x * blockDim.x + threadIdx.x;

        CuIntersection *its = &cuConstRendererParams.intersections[iid];
        

        CuEmitter *e = &cuConstRendererParams.emitters[0];
        

        CuRay *r = &cuConstRendererParams.queues1[iid];

        //if(iid > 30000 && iid < 30200) {
        //    printf("INTERSECTION: %d %d %d\n", iid, its->valid, r->id);
        //}

        if(!its->valid){ 
            r->valid = false;
            return;
        }
        
        hiprandState *rand = &cuConstRendererParams.randomStates[iid];
        float2 sample = squareSample(rand);
        // Generate the sample.
        float sampleX = (sample.x - 0.5);
        float sampleY = (sample.y - 0.5);

        
        float3 lpt = e->position + sampleX * e->dim_x + sampleY * e->dim_y;
        float3 d =  lpt - its->pt;
        float cosTheta = dot(d, e->direction);
        float sqDist = dot(d,d);
        float dist = sqrt(sqDist);
        r->d = d / dist;
        r->o = its->pt;
        float distToLight = dist;
        float pdf = sqDist / (e->area * abs(cosTheta));
        float fpdf = abs(dot(its->n, r->d))/ pdf;
        r->lightImportance = its->importance * make_float3(fpdf, fpdf, fpdf) * e->radiance;
        r->maxT = distToLight;
        r->importance = its->importance;
        r->sid = its->sid;
        r->light = its->light;
        r->id = its->id;

        // copied from its so that ray can easily duplicate value.
        r->n = its->n;
        r->wi = its->wi;
        r->t = its->t;
        r->valid = true;

        /*if(r->id > 30000 && r->id < 30200 && its->valid) {
            printf("INTERSECTION\n");
            printf("d: %f %f %f\n", r->d.x, r->d.y, r->d.z);
            printf("o: %f %f %f\n", r->o.x, r->o.y, r->o.z);
            printf("lightImportance: %f %f %f\n", r->lightImportance.x, r->lightImportance.y, r->lightImportance.z);
            printf("maxT: %f\n", r->maxT);
            printf("ID: %d\n", r->id);
            printf("SID: %d\n", r->sid);
            //printf("importance: %f %f %f\n", r->importance.x, r->importance.y, r->importance.z);
            //printf("its importance: %f %f %f\n", its->importance.x, its->importance.y, its->importance.z);
            printf("ID: %d SS: %f %f o: %f %f %f lpt: %f %f %f pt: %f %f %f t: %f d: %f %f %f its n: %f %f %f r->d: %f %f %f light: %f %f %f NEW: %d\n", r->id, r->ss.x, r->ss.y, r->o.x, r->o.y, r->o.z, lpt.x, lpt.y, lpt.z, its->pt.x, its->pt.y, its->pt.z, its->t, r->d.x, r->d.y, r->d.z, its->n.x, its->n.y, its->n.z, r->d.x, r->d.y, r->d.z, r->light.x, r->light.y, r->light.z, its->is_new);
            //printf("its n: %f %f %f\n", its->n.x, its->n.y, its->n.z);
        }*/
    }

    __global__ void kernelPrintLevelLists(int level, int total) {
        for(int i = 0; i < total; i++) {
            int nidx = cuConstRendererParams.levelIndices[level * LEVEL_INDEX_SIZE + i];
            printf("%d->%d\n", nidx, cuConstRendererParams.qCounts[nidx]);
        }
    }

    __global__ void kernelClearIntersections( ) {
        int iid = blockIdx.x * blockDim.x + threadIdx.x;
        for(int i = 0; i < MAX_INTERSECTIONS; i++) {
            CuIntersection *its = &cuConstRendererParams.multiIntersections[iid * MAX_INTERSECTIONS + i];
            its->valid = false;
        }

        cuConstRendererParams.intersections[iid].valid = false;
        cuConstRendererParams.minT[iid] = INFINITY;
        cuConstRendererParams.intersectionTokens[iid] = 0;
    }

    //__global__ void kernelClearIntersectionApparatus( ) {
    //    int iid = blockIdx.x * blockDim.x + threadIdx.x;
    //    cuConstRendererParams.minT[iid] = INFINITY;
    //    cuConstRendererParams.intersectionTokens[iid] = 0; 
    //}

    __global__ void kernelMergeIntersections( ) {
        int iid = blockIdx.x * blockDim.x + threadIdx.x;

        float t = INFINITY;
        CuIntersection best;
        for(int i = 0; i < MAX_INTERSECTIONS; i++) {
            CuIntersection *its = &cuConstRendererParams.multiIntersections[iid * MAX_INTERSECTIONS + i];
            //if(!its->valid) continue;
            if( its->valid && its->sort_t < t ) {
                //printf("VALID ITS: %d %f\n", iid, its->t);
                t = its->sort_t;
                best = *its;
            }
        }

        if(t != INFINITY)
            cuConstRendererParams.intersections[iid] = best;
    }

#define BSDF_DIFFUSE_MULTIPLIER 1.0
    // Generate secondary rays from the given intersections.
    __global__ void kernelProcessIntersections( ) {

        // For each element in intersection.
        // Check BSDF.
        // If 0 (Diffuse):
        // Randomly sample each intersection.
        // If 1 (Specular):
        // Find reflected ray.

        // Create new ray.
        // Compute importance for this ray.
        // Add to ray list at the same space as the intersection.

        int iid = blockIdx.x * blockDim.x + threadIdx.x;
        CuIntersection *its = &cuConstRendererParams.intersections[iid];
        CuRay *r = &cuConstRendererParams.queues1[0];
        if(!its->is_new) {
            // This intersection slot is stale. Ignore
            return;
        }

        // Compute a random sample. Hemispherical Random Sample.
        float sampleX;
        float sampleY;
        float sampleZ;

        float3 n = its->n;
        float3 dpdu; //TODO:Compute;
        float3 dpdv; //TODO:Compute;

        float dX = n.x * sampleZ + sampleX * dpdu.x + sampleY * dpdv.x;
        float dY = n.y * sampleZ + sampleX * dpdu.y + sampleY * dpdv.y;
        float dZ = n.z * sampleZ + sampleX * dpdu.z + sampleY * dpdv.z;

        float3 d = make_float3(dX, dY, dZ);
        float3 o = its->pt;
        r->d = d;
        r->o = o;

        int bsdfID = its->bsdf;
        CuBSDF *bsdf = &cuConstRendererParams.bsdfs[bsdfID];

        if(bsdf->fn == 0) {
            r->importance = its->importance * dot(r->d, its->n) * bsdf->albedo * BSDF_DIFFUSE_MULTIPLIER; // TODO: Compute with BSDF.
            r->light = its->light;
            r->lightImportance = make_float3(0, 0, 0);
            r->maxT = INFINITY;
            r->sid = its->sid;
            r->id = its->id;
        } else if(bsdf->fn == 1){
            // TODO: Implement specular stuff.
            r->importance = its->importance * dot(r->d, its->n); // TODO: Compute with BSDF.
            r->light = its->light;
            r->lightImportance = make_float3(0, 0, 0);
            r->maxT = INFINITY;
            r->sid = its->sid;
            r->id = its->id;
        }

        // Reset new bit so that we don't confuse intersections at a later stage.
        its->is_new = 0;
    }

    __global__ void kernelUpdateSSImage( ) {
        // For each element in intersection list.
        // Update the its.ss pixels using a reconstruction filter into
        // imageData.

        int width = cuConstRendererParams.imageWidth;
        int height = cuConstRendererParams.imageHeight;
        int sampleCount = cuConstRendererParams.sampleCount;

        int iid = blockIdx.x * blockDim.x + threadIdx.x;
        float4 *fx = &cuConstRendererParams.ssImageData[iid];
        CuIntersection *its = &cuConstRendererParams.intersections[iid];
            
        if(its->valid) {

            int x = static_cast<int>(its->ss.x);
            int y = static_cast<int>(its->ss.y);

            int sid = its->sid;

            //float4 *fx = &cuConstRendererParams.imageData[((y * height + x) * sampleCount + sid)];
            *fx = make_float4(its->light.x, its->light.y, its->light.z, 1.0);
            //*fx = make_float4(its->t / 5.0f, its->t / 5.0f, its->t / 5.0f, 1.0);
        } else {
            *fx = make_float4(0.0, 0.0, 0.0, 1.0);
        }
        
        
    }

    // Box Filter.
    // Soon change to Gaussian.
    __global__ void kernelReconstructImage( ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        int scount = cuConstRendererParams.sampleCount;

        float4 color = make_float4(0, 0, 0, 0);
        for(int i = 0; i < scount; i++) {
            float4 localColor = cuConstRendererParams.ssImageData[idx * scount + i];
            color += localColor / scount;
        }
        
        //printf("%d->%f %f %f %f\n", idx, color.x, color.y, color.z, color.w);
        cuConstRendererParams.imageData[idx] = color;
    }

    __global__ void kernelResetCounts( ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int width = cuConstRendererParams.imageWidth;
        int height = cuConstRendererParams.imageHeight;
        int sampleCount = cuConstRendererParams.sampleCount;
        
        int totalRays = width * height * sampleCount;

        if(idx == 0)
            cuConstRendererParams.qCounts[idx] = totalRays;
        else
            cuConstRendererParams.qCounts[idx] = 0;
        

    }

    // Intersection functions
    // Performs ray intersect on a single node.
    __device__ void rayIntersectSingle(int snode, int index, CuRay* inputQueue, CuRay* outputQueue) {

        // Mapping: Each block takes a subset of rays.

        // Each thread takes one ray.

        // Combined load of the data for the snode to shared memory.

        // Test all 16 child node BBoxes against every ray.

        // If hit:
        // (TODO) Add to the queue for that outlet.

        // If miss:
        // Leave it. 
        //printf("%d, %d\n", snode, index);
        bool is_leaf = (cuConstRendererParams.bvhSubTrees[snode].range != 0);
        int sampleCount = cuConstRendererParams.sampleCount;
        int imageWidth = cuConstRendererParams.imageWidth;
        int imageHeight = cuConstRendererParams.imageHeight;
        int maxRayCount = imageWidth * imageHeight * sampleCount;
        int rayCount = min(maxRayCount, cuConstRendererParams.qCounts[snode]);

        __shared__ int tindex;
        //printf("%d, %d\n", is_leaf, index);

        //return;



        //if(index >= rayCount) {
        //    return;
        //}

        //if(threadIdx.x == 0) {
        //    printf("SNODE Read: %d->%d\n", snode, snode * maxRayCount);
        //}

        //CuRay *r = &cuConstRendererParams.queues[maxRayCount * snode + index];
        //CuRay *raylist = &cuConstRendererParams.queues[maxRayCount * snode];
       

        __shared__ CuBVHSubTree subtree;
        //bool is_leaf = (cuConstRendererParams.bvhSubTrees[snode].range != 0);
        //printf("%d, %d\n", is_leaf, index);
        //if(r->valid && index < 200 && index < rayCount)
        //    printf("STREE: %d, %d: %d, %d valid: %d\n", is_leaf, index, rayCount, maxRayCount, r->valid);

        //if(is_leaf) {
        //printf("LEAFFF: %d, %d: %d, %d\n", is_leaf, index, rayCount, maxRayCount); 
        //}
        //return;
        // Combined load.
        int subindex = threadIdx.x;
        if(subindex < 1) {
            subtree.start = cuConstRendererParams.bvhSubTrees[snode].start;
            subtree.range = cuConstRendererParams.bvhSubTrees[snode].range;
            subtree.wOffset = cuConstRendererParams.bvhSubTrees[snode].wOffset; // Write offset. Given by kernelScanCounts();
            subtree.rOffset = cuConstRendererParams.bvhSubTrees[snode].rOffset; // Read offset. Set by previous kernelRayIntersectLevel();
        }


        //if(index == 0)
        //    printf("AFTER STREE: %d, %d\n", is_leaf, index);

        // TODO: Test of RAYS_PER_BLOCK being a power of 2.

        if(subindex < TREE_WIDTH) {
            subtree.outlets[subindex] = cuConstRendererParams.bvhSubTrees[snode].outlets[subindex];
            subtree.minl[subindex + 0] = cuConstRendererParams.bvhSubTrees[snode].minl[subindex + 0];
            subtree.maxl[subindex + 0] = cuConstRendererParams.bvhSubTrees[snode].maxl[subindex + 0];
            
            
            if(subtree.outlets[subindex] != (uint64_t) -1){
                //if(subtree.outlets[subindex] > 600)

                cuConstRendererParams.bvhSubTrees[subtree.outlets[subindex]].rOffset = subtree.wOffset + subindex * rayCount; // Read offset for next level is based on write offset for this level.
            }
        } 
        

        __syncthreads();
        
        int rOffset = (snode == 0) ? 0 : subtree.rOffset;// + subindex * rayCount;
        int wOffset = subtree.wOffset;
        //outputOffsets[linearIndex] = ;
        
        CuRay *r = &inputQueue[rOffset + index];
        CuRay *raylist = &inputQueue[rOffset];
        
        //if(r->id == 528) {
        //if(snode == 294 && index < rayCount) {
        //    printf("%d: FOUND ID %d at %d+%d\n", snode, r->id, rOffset, index);
       // }
        
        //if(index == 0)
        //    printf("STREE: node: %d, %d, %d: %d, %d, %d, %d\n", snode, is_leaf, index, rayCount, maxRayCount, rOffset, wOffset);

        if(!is_leaf) {

            __shared__ uint _outlets[TREE_WIDTH * RAYS_PER_BLOCK];
            __shared__  uint _c_outlets[TREE_WIDTH * RAYS_PER_BLOCK];
            //__shared__ uint _compacter[TREE_WIDTH * RAYS_PER_BLOCK];
            __shared__ uint _c_qid[2 * TREE_WIDTH * RAYS_PER_BLOCK];
            //__shared__ uint _scratch[TREE_WIDTH * RAYS_PER_BLOCK];
            //if(index == 0)
            //    printf("STREE: %d, %d\n", is_leaf, index);
            //int subindex = index % (RAYS_PER_BLOCK);

            float minT = cuConstRendererParams.minT[r->id];
            /*if(index == 2100) {
              printf("SubTree: Start->%lu\n", subtree.start);
              printf("SubTree: Range->%lu\n", subtree.range);
              for(int i = 0; i < TREE_WIDTH; i++)
              printf("SubTree: Outlets[%d]->%lu\n", i, subtree.outlets[i]);
              for(int i = 0; i < TREE_WIDTH; i++)
              printf("SubTree: Min[%d]->%f %f %f\n", i, subtree.minl[i].x, subtree.minl[i].y, subtree.minl[i].z);
              for(int i = 0; i < TREE_WIDTH; i++)
              printf("SubTree: Max[%d]->%f %f %f\n", i, subtree.maxl[i].x, subtree.maxl[i].y, subtree.maxl[i].z);

            //printf("SUBTREE_TEST: %d\n",cuConstRendererParams.bvhSubTrees[0].outlets[0]);
                printf("\n");
            }*/

            for(int i = 0; i < TREE_WIDTH; i++) {
                _outlets[i * RAYS_PER_BLOCK + subindex] = 0; 
            }
            //return;
            __syncthreads();

            if(index < rayCount && r->valid) {

                for(int i = 0; i < TREE_WIDTH; i++) {
                    // Intersect the rays here.
                    //if(r->id == 33024) {
                        //printf("OUTLET[%d]=%lu\n", i, subtree.outlets[i]);
                    //}
                    if(subtree.outlets[i] == (uint64_t)-1) continue;

                    float t = intersectBBox(r->o, r->d, subtree.minl[i], subtree.maxl[i]);
                    
                    if( t >= 0 && t <= minT ) {
                        // If intersected, place a mark.
                        _outlets[i * RAYS_PER_BLOCK + subindex] = 1;
                        //printf("RAY %d,%d,%d: %f\n", i, RAYS_PER_BLOCK, subindex, t);
                    } else {
                        _outlets[i * RAYS_PER_BLOCK + subindex] = 0;
                    }

                }

           }

                //return;
                // Perform compaction on every 512x:(512x + 512)
                //_assignments[threadIdx.x] = assignments[(blockIdx.x << sizelog) + threadIdx.x];
                //if(blockIdx.x == 1000 && _assignments[threadIdx.x]){// && _assignments[(blockIdx.x << sizelog) + threadIdx.x]) {
                //    printf("%d: %u\n", threadIdx.x, blockIdx.x << sizelog);
                //}
                //_compacter[threadIdx.x] = compacter[blockIdx.x * size + threadIdx.x];
                //if(threadIdx.x % 10 == 0)
                //    printf("hello\n");
                // Wait for sync.
                __syncthreads();
                //if(index == 2100)
                //    printf("EXSCAN: %d, %d, %d, %d\n", RAYS_PER_BLOCK, subindex, TREE_WIDTH, blockDim.x);

                //if(subindex > RAYS_PER_BLOCK) {
                //    printf("EXSCAN: %d, %d, %d, %d\n", RAYS_PER_BLOCK, subindex, TREE_WIDTH, blockDim.x); 
                //}

                //return;
                //sharedMemExclusiveScan(thread, _assignments, _compacter[sliceIdx * size], &scratch[sliceIdx * size], size);
                for(int i = 0; i < TREE_WIDTH; i++) {
                    //if(subtree.outlets[i] == (uint64_t)-1) continue;
                    
                    sharedMemExclusiveScan(subindex, &_outlets[i * RAYS_PER_BLOCK], &_c_outlets[i * RAYS_PER_BLOCK], &_c_qid[i * 2 * RAYS_PER_BLOCK], RAYS_PER_BLOCK);
                }
                __syncthreads();

                /*if(subindex == 0) {
                    if(_c_outlets[RAYS_PER_BLOCK - 1] % 4 != 0) {
                        printf("BLOCK: %d Request: %d IDX: %d RC: %d\n", blockIdx.x, _c_outlets[RAYS_PER_BLOCK - 1], index, rayCount);
                        for(int i = 0; i < RAYS_PER_BLOCK; i+=4)
                            printf("%d:%d%d%d%d ", i, _outlets[i], _outlets[i+1], _outlets[i+2], _outlets[i+3]);
                        printf("\n");
                        //for(int i = 0; i < RAYS_PER_BLOCK; i++)
                        //    printf("%d:%d ", i, _c_outlets[i]);
                        //printf("\n");
                    } 

                }*/
                //return;
                // Rearrange.
                for(int i = 0; i < TREE_WIDTH; i++) {
                    if(subindex >= RAYS_PER_BLOCK) 
                        continue;

                    uint k0 = _c_outlets[i * RAYS_PER_BLOCK + subindex + 0];
                    uint k1 = _c_outlets[i * RAYS_PER_BLOCK + subindex + 1]; 
                    //printf("%d,%d->%d,%d - %d\n", i, subindex, k0, k1, _c_outlets[(i+1) * RAYS_PER_BLOCK - 1]);
                    if(subindex != RAYS_PER_BLOCK - 1) {
                        if((k0 + 1) == k1) {
                            _c_qid[i * RAYS_PER_BLOCK + k0] = index;
                            //printf("%d->%d\n", k0, index);
                        }
                    } else {
                        if(_outlets[i * RAYS_PER_BLOCK + subindex])
                            _c_qid[i * RAYS_PER_BLOCK + k0] = index;
                    }
                }

                __syncthreads();
                for(int i = 0; i < TREE_WIDTH; i++) {
                    uint64_t target = subtree.outlets[i];

                    __syncthreads();
                    // Atomic grab.
                    //if (raylist[rayid].id == 1536) {
                    int numRays = _c_outlets[(i+1) * RAYS_PER_BLOCK - 1] + _outlets[(i+1) * RAYS_PER_BLOCK - 1];
                    if(subindex == 0 && (target != (uint64_t)-1)){
                        tindex = atomicAdd(&cuConstRendererParams.qCounts[target], numRays);
                    }
                    __syncthreads();

                    if((target != (uint64_t)-1) && subindex < numRays) { 
                        int rayid = _c_qid[i * RAYS_PER_BLOCK + subindex];
                        //cuConstRendererParams.queues[maxRayCount * target + tindex + subindex] = raylist[rayid];
                        //if(wOffset + i * rayCount + tindex + subindex > 2097152) { printf("Illegal access: %d = %d+%d+%d+%d \n", wOffset + i * rayCount + tindex + subindex, wOffset, i * rayCount, tindex, subindex); }
                        outputQueue[(wOffset + i * rayCount) + tindex + subindex] = raylist[rayid];
                    }

                    //if( subindex == 0 && numRays != 0 ) {
                    //  printf("SNODE: %d TINDEX--------------(B:%dx%d) %d+%d (tot:%d) %lu->%d\n", snode, blockIdx.x, blockDim.x, subindex, tindex, numRays, target, wOffset + i * rayCount);
                    //}
                    //}

                    /*if(raylist[_c_qid[i * RAYS_PER_BLOCK + subindex]].id == 65664 && (target != (uint64_t)-1) && (subindex < numRays) ){
                    //subtree.
                    //CuBVHSubTree subtree = cuConstRendererParams.bvhSubTrees[snode];
                    int rayid = _c_qid[i * RAYS_PER_BLOCK + subindex];
                    float t = intersectBBox(raylist[rayid].o, raylist[rayid].d, subtree.minl[i], subtree.maxl[i]);
                    printf("Node: %d %lu->%d+%d,ID:%d\n", snode, target, cuConstRendererParams.qCounts[target], numRays, raylist[rayid].id);
                    printf("Target: %lu->%d Raycount: %d\n", target, wOffset + i * rayCount, rayCount);
                    printf("index:%d-i:%d \nSource Node: %d Node:%lu->Loc:%d+%d, RAYID: %d/%d \n MAX_INDICES: %d\nID:%d\n t:%f \nO:%f %f %f \n D:%f %f %f \nmin: %f %f %f\nmax: %f %f %f\n\n\n", index, i, snode, target, wOffset + i * rayCount, tindex + subindex, rayid, rayCount, numRays, raylist[rayid].id, t, raylist[rayid].o.x, raylist[rayid].o.y, raylist[rayid].o.z, raylist[rayid].d.x, raylist[rayid].d.y, raylist[rayid].d.z, subtree.minl[i].x, subtree.minl[i].y, subtree.minl[i].z, subtree.maxl[i].x, subtree.maxl[i].y, subtree.maxl[i].z);


                    printf("tindex: %lu, %d, %lu, %f\n", maxRayCount * target, tindex + subindex, maxRayCount * target + tindex + subindex, t);
                    //printf("Ntest: %f\n\n", t);
                    }*/

                    //__syncthreads();


                }

            } else {

                __shared__ CuTriangle _triangles[MAX_TRIANGLES];
                int num_triangles = cuConstRendererParams.bvhSubTrees[snode].range;

                //return;
                // Copy to shared memory

                //int subindex = threadIdx.x;

                if(subindex < num_triangles) {
                    _triangles[subindex] = cuConstRendererParams.triangles[cuConstRendererParams.bvhSubTrees[snode].start + subindex];
                }

                //return;

                __syncthreads();

                float t = INFINITY;

                CuTriangle tri;
                // Perform triangle intersect.
                if(index < rayCount && r->valid) {
                    for(int i = 0; i < num_triangles; i++) {
                        int start = cuConstRendererParams.bvhSubTrees[snode].start;
                        //if(subindex == 0 && (start + i > 1733) && (start + i < 1745)){
                        float thist = intersectRayTriangle(_triangles[i].a, _triangles[i].b, _triangles[i].c, r->o, r->d);
                        if(thist < t && thist >= 0){
                            //if(thist >= 0){
                            t = thist;
                            tri = _triangles[i];
                            //}
                        }

                    }
                }


                //int x = static_cast<int>(r->ss.x);
                //int y = static_cast<int>(r->ss.y);

                //int sid = r->sid;

                int imageWidth = cuConstRendererParams.imageWidth;
                int imageHeight = cuConstRendererParams.imageHeight;
                //int iid = ((y * imageHeight + x) * sampleCount + sid);

                int iid = r->id;


                if(iid >= imageWidth * imageHeight * SAMPLES_PER_PIXEL) 
                    printf("IID exceeds total samples.%d\n", iid);

                //float4 *fx = &imageData[((y * height + x) * sampleCount + sid)];
                //CuIntersection *ac_its = &cuConstRendererParams.intersections[((y * imageHeight+ x) * sampleCount + sid)];
                float tmin = cuConstRendererParams.minT[iid];
                uint *tokens = &cuConstRendererParams.intersectionTokens[iid];
                CuIntersection its;

                //if(atomicMin(tmin, t) != t) {
                if(t == INFINITY || tmin < t) {
                    return;
                }

                cuConstRendererParams.minT[iid] = t;

                // Take a token.
                int token = atomicAdd(tokens, 1);
                bool direct_light = !(r->lightImportance == make_float3(0.0, 0.0, 0.0));
                if(!direct_light) {
                    // Overwrite the intersection.
                    its.t = t;
                    its.sort_t = t;
                    its.pt = (r->o + r->d * t);// + r->n * 1e-4;
                    //its->lightImportance = r->lightImportance;
                    its.light = r->light;
                    its.importance = r->importance;

                    //float3 n = normalize(cross(tri.a - tri.b, tri.b - tri.c));
                    //its->n = ((dot(n, r->d) < 0) ? -1 : 1) * n;

                    // Compute barycentric coordinates.
                    float total = length(cross(tri.a - tri.b, tri.b - tri.c));
                    //float bC = 0.0;
                    //float bA = 0.0;
                    //float bB = 0.0;

                    float bC = length(cross(tri.a - its.pt, tri.b - its.pt)) / total;
                    float bA = length(cross(tri.b - its.pt, tri.c - its.pt)) / total;
                    float bB = length(cross(tri.c - its.pt, tri.a - its.pt)) / total;
                    its.n = bA * tri.n0 + bB * tri.n1 + bC * tri.n2;
                    its.n = (its.n) * (dot(its.n, r->d) < 0 ? 1 : -1);
                    
                    its.pt += its.n * 1e-4;

                    // Make 2 more axes.
                    float3 ax = normalize(cross(make_float3(0.1, 0.1, 1), its.n));
                    float3 ay = normalize(cross(ax, its.n));
                    its.wi = normalize(make_float3(dot(ax, r->d), dot(ay, r->d), dot(its.n, r->d)));

                    its.ss = r->ss;
                    its.sid = r->sid;
                    its.id = r->id;
                    its.bsdf = tri.bsdf;
                    its.is_new = 1;
                    its.valid = true;
                } else {
                    // If direct light estimate, then only estimate the light at this point.
                    

                    its.n = r->n;
                    its.wi = r->wi;
                    its.ss = r->ss;
                    its.sid = r->sid;
                    its.id = r->id;
                    its.importance = r->importance;
                    its.pt = r->o;//make_float3(r->maxT, r->maxT, r->maxT);//TODO: CHANGED CHANGE THIS BACK TODO TODO TODO TODO
                    its.t = r->t; //TODO: CHANGED CHANGE THIS BACK TODO TODO TODO TODO
                    its.sort_t = t;
                    its.bsdf = tri.bsdf;
                    its.light = r->light + ((t > r->maxT - 1e-4) ? (r->lightImportance) : make_float3(0.0)); // TODO: Make update.
                    its.is_new = 2;
                    its.valid = true;
                    //if(r->id > 30000 && r->id < 30200 && r->valid) {
                    //    printf("Intersection ID: %d %d\n", its.id, its.is_new);
                    //}
                    
                }

                // write this intersection out.
                cuConstRendererParams.multiIntersections[iid * MAX_INTERSECTIONS + token] = its;
            }


            }

            __global__ void kernelSetupRandomSeeds(){
            	int idx = blockIdx.x * blockDim.x + threadIdx.x;
            	hiprand_init(seed, idx, 0, &cuConstRendererParams.randomStates[idx]);
            }

            __global__ void kernelRayIntersectSingle(int snode) {
                int index = blockDim.x * blockIdx.x + threadIdx.x;
                //if(index == 0) {
                //    printf("INDEX: %d\n\n", index);
                //}
                rayIntersectSingle(snode, index, cuConstRendererParams.queues1, cuConstRendererParams.queues2);
            }

            __global__ void kernelPrintQueueCounts() {
                //printf("%u->%u\n", threadIdx.x, cuConstRendererParams.qCounts[threadIdx.x]);
            }

            __global__ void kernelScanCounts(int level) {
                int idx = threadIdx.x;
                __shared__ uint inputCounts[512];
                __shared__ uint outputCounts[512];
                __shared__ uint spare[1024];
                
                int nodeIdx = cuConstRendererParams.levelIndices[level * LEVEL_INDEX_SIZE + idx];
                inputCounts[idx] = cuConstRendererParams.qCounts[nodeIdx];
                __syncthreads();
                sharedMemExclusiveScan(idx, &inputCounts[0], &outputCounts[0], &spare[0], 512);
                __syncthreads();

                cuConstRendererParams.bvhSubTrees[nodeIdx].wOffset = outputCounts[idx] * TREE_WIDTH; // Provide space for all branches (worst-case)
            }

            // Intersection function.
            // Performs ray intersection on a full level.
            __global__ void kernelRayIntersectLevel(int level) {
                // Mapping: Each block takes a subset of rays split for that level.
                // Use blockID bits to split the rays to each snode in the level.
                // Each block only works on one particular snode.

                // Each thread takes one ray.

                // Combined load of the data for the snode to shared memory.

                // Test all 16 child node BBoxes against every ray.

                // If hit:
                // (TODO) Add to the queue for that outlet.

                // If miss:
                // Leave it.

                // Use a table to compute the nodes at this level.

                // Compute the queue index.
                //int levelIndex = (blockIdx.x * blockDim.x) >> QUEUE_LENGTH_LOG2;
                int imageWidth = cuConstRendererParams.imageWidth;
                int imageHeight = cuConstRendererParams.imageHeight;
                int sampleCount = cuConstRendererParams.sampleCount;

                int levelIndex = (blockIdx.x * blockDim.x) / (imageWidth * imageHeight * sampleCount);

                int nodeIndex = cuConstRendererParams.levelIndices[level * LEVEL_INDEX_SIZE + levelIndex];
                int rayIndex = (blockIdx.x * blockDim.x + threadIdx.x) % (imageWidth * imageHeight * sampleCount);
                

                //if(rayIndex == 0){
                //    printf("At %d\n", nodeIndex);
                //}
                //if(level < 4) { 
                int rayCount = cuConstRendererParams.qCounts[nodeIndex];

                __shared__ bool active;

                if(threadIdx.x == 0) {
                    active = (rayIndex <= rayCount);
                }
                
                __syncthreads();
                
                if (active) {
                    if(level % 2 == 0) {
                        rayIntersectSingle(nodeIndex, rayIndex, cuConstRendererParams.queues1, cuConstRendererParams.queues2);
                    } else {
                        rayIntersectSingle(nodeIndex, rayIndex, cuConstRendererParams.queues2, cuConstRendererParams.queues1); 
                    }
                }
                //}
            }


            //#define NUM_CIRCLES_PER_BLOCK 1024
#define NUM_LAYERS 3

            //#define LOG_CIRCLES_PER_BLOCK 13
            CudaRenderer::CudaRenderer() {
                image = NULL;
                deviceBSDFs = NULL;
                deviceEmitters = NULL;
                deviceBVHSubTrees = NULL;
                deviceTriangles = NULL;
                deviceRays1 = NULL;
                deviceRays2 = NULL;
                deviceIntersections = NULL;
                deviceImageData = NULL;
                deviceSSImageData = NULL;
                deviceLevelIndices = NULL;
            }

            CudaRenderer::~CudaRenderer() {

                if (image) {
                    delete image;
                }

                if (bsdfs.size() > 0) {
                    bsdfs.clear();
                    emitters.clear();
                    subtrees.clear();
                    triangles.clear();
                    delete [] levelIndices;
                }

                if (deviceBSDFs) {
                    hipFree(deviceBSDFs);
                    hipFree(deviceEmitters);
                    hipFree(deviceBVHSubTrees);
                    hipFree(deviceTriangles);   
                    hipFree(deviceRays1);
                    hipFree(deviceRays2);
                    hipFree(deviceIntersections);
                    hipFree(deviceSSImageData);
                    hipFree(deviceImageData);
                    hipFree(deviceLevelIndices);

                }
            }

            const Image* CudaRenderer::getImage() {

                // Need to copy contents of the rendered image from device memory
                // before we expose the Image object to the caller

                printf("Copying image data from device\n");

                hipMemcpy(image->data,
                        deviceImageData,
                        sizeof(float) * 4 * image->width * image->height,
                        hipMemcpyDeviceToHost);

                return image;
            }

            DynamicScene::Scene* CudaRenderer::loadFromSceneInfo(Collada::SceneInfo* sceneInfo) {

                vector<Collada::Node> &nodes = sceneInfo->nodes;
                vector<DynamicScene::SceneLight *> lights;
                vector<DynamicScene::SceneObject *> objects;

                // save camera position to update camera control later
                Collada::CameraInfo *c;
                Vector3D c_pos = Vector3D();
                Vector3D c_dir = Vector3D();

                int len = nodes.size();
                for (int i = 0; i < len; i++) {
                    Collada::Node &node = nodes[i];
                    Collada::Instance *instance = node.instance;
                    const Matrix4x4 &transform = node.transform;

                    switch (instance->type) {
                        case Collada::Instance::CAMERA: {
                                                            c = static_cast<Collada::CameraInfo *>(instance);
                                                            c_pos = (transform * Vector4D(c_pos, 1)).to3D();
                                                            c_dir = (transform * Vector4D(c->view_dir, 1)).to3D().unit();
                                                            std::cout << "Camera parameters: " << std::endl;
                                                            this->c_lookAt = -c_dir;
                                                            this->c_origin = c_pos + Vector3D(0, 0.5, 0);
                                                            Vector3D acup(0.0f, 1.0f, 0.0f);
                                                            this->c_left = cross(acup, c_dir).unit();
                                                            this->c_up = cross(this->c_left, c_dir).unit();
                                                            std::cout << "lookAt: " << this->c_lookAt << std::endl;
                                                            std::cout << "origin: " << this->c_origin << std::endl;
                                                            std::cout << "left: " << this->c_left << std::endl;
                                                            std::cout << "up: " << this->c_up << std::endl;

                                                            init_camera(*c, transform);
                                                            break;
                                                        }
                        case Collada::Instance::LIGHT: {
                                                           lights.push_back(
                                                                   init_light(static_cast<Collada::LightInfo &>(*instance), transform));
                                                           break;
                                                       }
                        case Collada::Instance::SPHERE:
                                                       objects.push_back(
                                                               init_sphere(static_cast<Collada::SphereInfo &>(*instance), transform));
                                                       break;
                        case Collada::Instance::POLYMESH:
                                                       objects.push_back(
                                                               init_polymesh(static_cast<Collada::PolymeshInfo &>(*instance), transform));
                                                       break;
                        case Collada::Instance::MATERIAL:
                                                       //init_material(static_cast<Collada::MaterialInfo &>(*instance));
                                                       std::cout << "Unable to handle material.\n" << std::endl;
                                                       break;
                    }
                }

                // TODO: TEmporarily disabled this to test environment lights.
                if (lights.size() == 0) {  // no lights, default use ambient_light
                    Collada::LightInfo default_light = Collada::LightInfo();
                    lights.push_back(new DynamicScene::AmbientLight(default_light));
                }
                DynamicScene::Scene* scene = new DynamicScene::Scene(objects, lights);

                const BBox &bbox = scene->get_bbox();
                if (!bbox.empty()) {
                    //Vector3D target = bbox.centroid();
                    //canonical_view_distance = bbox.extent.norm() / 2 * 1.5;

                    //double view_distance = canonical_view_distance * 2;
                    //double min_view_distance = canonical_view_distance / 10.0;
                    //double max_view_distance = canonical_view_distance * 20.0;

                    //canonicalCamera.place(target, acos(c_dir.y), atan2(c_dir.x, c_dir.z),
                    //                      view_distance, min_view_distance, max_view_distance);

                    //camera.place(target, acos(c_dir.y), atan2(c_dir.x, c_dir.z), view_distance,
                    //             min_view_distance, max_view_distance);

                    //set_scroll_rate();
                }

                // set default draw styles for meshEdit -
                //scene->set_draw_styles(&defaultStyle, &hoverStyle, &selectStyle);

                // cerr << "==================================" << endl;
                // cerr << "CAMERA" << endl;
                // cerr << "      hFov: " << camera.hFov << endl;
                // cerr << "      vFov: " << camera.vFov << endl;
                // cerr << "        ar: " << camera.ar << endl;
                // cerr << "     nClip: " << camera.nClip << endl;
                // cerr << "     fClip: " << camera.fClip << endl;
                // cerr << "       pos: " << camera.pos << endl;
                // cerr << " targetPos: " << camera.targetPos << endl;
                // cerr << "       phi: " << camera.phi << endl;
                // cerr << "     theta: " << camera.theta << endl;
                // cerr << "         r: " << camera.r << endl;
                // cerr << "      minR: " << camera.minR << endl;
                // cerr << "      maxR: " << camera.maxR << endl;
                // cerr << "       c2w: " << camera.c2w << endl;
                // cerr << "   screenW: " << camera.screenW << endl;
                // cerr << "   screenH: " << camera.screenH << endl;
                // cerr << "screenDist: " << camera.screenDist<< endl;
                // cerr << "==================================" << endl;
                return scene;
            }

            void CudaRenderer::loadScene(std::string sceneFilePath) {
                //sceneName = scene;
                printf("Loading scene: %s\n", sceneFilePath.c_str()); 
                Collada::SceneInfo* sceneInfo = new Collada::SceneInfo();
                if (Collada::ColladaParser::load(sceneFilePath.c_str(), sceneInfo) < 0) {
                    printf("Error: parsing failed!\n");
                    delete sceneInfo;
                    exit(0);
                }

                DynamicScene::Scene* dscene = this->loadFromSceneInfo(sceneInfo);
                StaticScene::Scene* scene = dscene->get_static_scene();

                std::vector<StaticScene::Primitive *> primitives;
                for (StaticScene::SceneObject *obj : scene->objects) {
                    const vector<StaticScene::Primitive *> &obj_prims = obj->get_primitives();
                    primitives.reserve(primitives.size() + obj_prims.size());
                    primitives.insert(primitives.end(), obj_prims.begin(), obj_prims.end());
                }

                //std::vector<CuTriangle> cuts;

                // Add BSDFs
                // TODO: Make this automated soon.
                //std::vector<CuBSDF> bsdfs;
                CuBSDF b;
                b.albedo = make_float3(0.6, 0.6, 0.6);
                b.fn = 0;
                b.nu = 0;

                bsdfs.push_back(b);

                // Add Emitters
                // TODO: Make this automated soon.
                if(scene->lights.size() != 1) {
                    std::cout << "Error: Too many lights: " << scene->lights.size() << ". Can only handle one for now.\n";
                    exit(0);
                }

                CuEmitter e;
                auto l = scene->lights[0];
                auto al = reinterpret_cast<StaticScene::AreaLight*>(l);

                std::cout << "AreaLight: " << al->position << " ,\n" << al->direction << " ,\n" << al->dim_x << " ,\n" << al->dim_y << std::endl;
                std::cout << al->radiance << std::endl;

                e.position = v2f3(al->position);
                e.direction = v2f3(al->direction);
                e.dim_x = v2f3(al->dim_x);
                e.dim_y = v2f3(al->dim_y);
                e.radiance = make_float3(al->radiance.r, al->radiance.g, al->radiance.b);
                e.area = length(e.dim_x) * length(e.dim_y);
                std::cout << "Area: " << e.area;
                //std::vector<CuEmitter> emitters;
                emitters.push_back(e);


                auto bvh = new StaticScene::BVHAccel(primitives);
                
                std::cout << "Primitives loaded: " << primitives.size() << std::endl;
                for(auto prim : bvh->getSortedPrimitives()) {
                    StaticScene::Triangle* t = reinterpret_cast<StaticScene::Triangle*>(prim); 
                    Vector3D v0,v1,v2;
                    t->positions(v0, v1, v2);
                    Vector3D n0,n1,n2;
                    t->normals(n0, n1, n2);

                    CuTriangle ct;
                    ct.a = make_float3(v0.x, v0.y, v0.z);
                    ct.b = make_float3(v1.x, v1.y, v1.z);
                    ct.c = make_float3(v2.x, v2.y, v2.z);

                    ct.n0 = make_float3(n0.x, n0.y, n0.z);
                    ct.n1 = make_float3(n1.x, n1.y, n1.z);
                    ct.n2 = make_float3(n2.x, n2.y, n2.z);

                    ct.bsdf = 0;
                    ct.emit = -1; 

                    triangles.push_back(ct);

                    //std::cout << "Primitive: " << v0 << " " << v1 << " " << v2 << std::endl;
                }
                
                int scount;
                //this->subtrees = bvh->compressedTree();
                auto tmp_subtrees = bvh->compactedTree(); // Tree compaction system to make the tree smaller.

                std::vector<StaticScene::C_BVHSubTree> *tree = new std::vector<StaticScene::C_BVHSubTree>();
                //std::vector<int> *levelCounts = new std::vector<int>();

                this->levelIndices = (int*) malloc(sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS);
                tmp_subtrees->compress(tree, this->levelIndices, LEVEL_INDEX_SIZE, &levelCounts, 0, MAX_LEVELS); // Compressed subtree.

                int curr = 0;
                for(int t = 0; t < tree->size(); t ++) {
                    auto entry = tree->at(t);
                    //if(entry.range != 0) continue;
                    //std::cout << "BVHSubTree entry " << curr++ << std::endl; 

                    //std::cout << "start: " << entry.start << std::endl;
                    //std::cout << "range: " << entry.range << std::endl;

                    CuBVHSubTree cutree;
                    cutree.start = entry.start;
                    cutree.range = entry.range;

                    for(int i = 0; i < TREE_WIDTH; i++){
                        //std::cout << "min " << i << ": " << entry.min[i] << std::endl;
                        //std::cout << "max " << i << ": " << entry.max[i] << std::endl;
                        //std::cout << "outlet " << i << ": " << entry.outlets[i] << std::endl;

                        cutree.minl[i] = v2f3(entry.min[i]);
                        cutree.maxl[i] = v2f3(entry.max[i]);
                        cutree.outlets[i] = entry.outlets[i];
                    }
                    this->subtrees.push_back(cutree);
                }

                std::cout << "\nLevel Profile\n";
                for(int t = 0; t < levelCounts.size(); t++) {
                    std::cout << t << "->" << levelCounts.at(t) << std::endl;
                }
                std::cout << std::endl;
                for(int i = 0; i < levelCounts.size(); i++) {
                    int sz = levelCounts.at(i);
                    for(int j = 0; j < sz; j++) {
                        printf("%d,",levelIndices[LEVEL_INDEX_SIZE * i + j]);
                    }
                    printf("\n");
                }

            }

            void CudaRenderer::setup() {

                int deviceCount = 0;
                bool isFastGPU = false;
                std::string name;
                hipError_t err = hipGetDeviceCount(&deviceCount);

                printf("---------------------------------------------------------\n");
                printf("Initializing CUDA for CudaRenderer\n");
                printf("Found %d CUDA devices\n", deviceCount);

                for (int i=0; i<deviceCount; i++) {
                    hipDeviceProp_t deviceProps;
                    hipGetDeviceProperties(&deviceProps, i);
                    name = deviceProps.name;
                    if (name.compare("GeForce GTX 1040") == 0)
                    {
                        isFastGPU = true;
                    }

                    printf("Device %d: %s\n", i, deviceProps.name);
                    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
                    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
                    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
                }
                printf("---------------------------------------------------------\n");
                if (!isFastGPU)
                {
                    printf("WARNING: "
                            "You're not running on a fast GPU, please consider using "
                            "NVIDIA GTX 480, 670 or 780.\n");
                    printf("---------------------------------------------------------\n");
                }

                // By this time the scene should be loaded.  Now copy all the key
                // data structures into device memory so they are accessible to
                // CUDA kernels
                //
                // See the CUDA Programmer's Guide for descriptions of
                // hipMalloc and hipMemcpy
                // Compute Pow2 and Log2 versions of numCircles, this is important for the compaction algorithm,
                // which only works with powers of 2/


                // TODO: WARN: Temporary override.
                //numCircles = 1024;

                int numRays = SAMPLES_PER_PIXEL * image->width * image->height;
                int queueSize = numRays * TREE_WIDTH * 2;
                
                std::cout << "Queue Size: " << queueSize << std::endl;
                std::cout << "\nDevice Allocation \n";
                std::cout << "BSDFS: " << bsdfs.size() << std::endl;
                std::cout << "Emitters: " << emitters.size() << std::endl;
                std::cout << "Triangles: " << triangles.size() << std::endl;
                std::cout << "BVHSubTrees: " << subtrees.size() << std::endl;
                
                std::cout << "BVH SIZE: " << sizeof(CuBVHSubTree) * subtrees.size() << std::endl;

                size_t total = 0;
                total += sizeof(CuBSDF) * bsdfs.size();
                total += sizeof(CuEmitter) * emitters.size();
                total += sizeof(CuTriangle) * triangles.size();
                total += sizeof(CuBVHSubTree) * subtrees.size();
                total += sizeof(CuRay) * queueSize;
                total += sizeof(CuIntersection) * numRays;
                total += sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS; 
                total += sizeof(float) * 4 * image->width * image->height * SAMPLES_PER_PIXEL;
                total += sizeof(float) * 4 * image->width * image->height;
                total += sizeof(uint) * subtrees.size();
                total += sizeof(float) * numRays;
                total += sizeof(uint) * numRays;
                total += sizeof(CuIntersection) * MAX_INTERSECTIONS * numRays;
                
                std::cout << "Total memory allocation: " << total / 1000000 << " MB" << std::endl;
                std::cout << "Device rays: " << sizeof(CuRay) * queueSize / 1000000 << " MB" << std::endl;

                auto ok = hipMalloc(&deviceBSDFs, sizeof(CuBSDF) * bsdfs.size());
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceEmitters, sizeof(CuEmitter) * emitters.size());
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceTriangles, sizeof(CuTriangle) * triangles.size());
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceBVHSubTrees, sizeof(CuBVHSubTree) * subtrees.size());
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceRays1, sizeof(CuRay) * queueSize);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceRays2, sizeof(CuRay) * queueSize);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceIntersections, sizeof(CuIntersection) * numRays);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceLevelIndices, sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS); 
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceSSImageData, sizeof(float) * 4 * image->width * image->height * SAMPLES_PER_PIXEL);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceImageData, sizeof(float) * 4 * image->width * image->height);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceQueueCounts, sizeof(uint) * subtrees.size());
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceMinT, sizeof(float) * numRays);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceIntersectionTokens, sizeof(uint) * numRays);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMalloc(&deviceMultiIntersections, sizeof(CuIntersection) * MAX_INTERSECTIONS * numRays);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}

                //hipMalloc(&deviceQueueOffsets1, sizeof(uint) * subtrees.size());
                //hipMalloc(&deviceQueueOffsets2, sizeof(uint) * subtrees.size());

                ok = hipMalloc(&deviceRandomStates, sizeof(hiprandState) * numRays);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}

                //int* hcounts = reinterpret_cast<int*>(calloc((image->width >> KWIDTH) * (image->height >> KWIDTH), sizeof(int)));

                //hipMemcpy(counts, hcounts, sizeof(int) * (image->width >> KWIDTH) * (image->height >> KWIDTH), hipMemcpyHostToDevice);
                //hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
                //hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
                //hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
                //hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);
                std::vector<uint> qcounts;
                qcounts.push_back(numRays);
                for(int i = 1; i < subtrees.size(); i++ ){
                    qcounts.push_back(0);
                }


                ok = hipMemcpy(deviceBSDFs, &bsdfs[0], sizeof(CuBSDF) * bsdfs.size(), hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMemcpy(deviceEmitters, &emitters[0], sizeof(CuEmitter) * emitters.size(), hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMemcpy(deviceTriangles, &triangles[0], sizeof(CuTriangle) * triangles.size(), hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMemcpy(deviceBVHSubTrees, &subtrees[0], sizeof(CuBVHSubTree) * subtrees.size(), hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMemcpy(deviceQueueCounts, &qcounts[0], sizeof(uint) * subtrees.size(), hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}
                ok = hipMemcpy(deviceLevelIndices, levelIndices, sizeof(int) * LEVEL_INDEX_SIZE * MAX_LEVELS, hipMemcpyHostToDevice);
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}

                printf("CuTest: %d\n", subtrees[0].outlets[0]);

                // Initialize parameters in constant memory.  We didn't talk about
                // constant memory in class, but the use of read-only constant
                // memory here is an optimization over just sticking these values
                // in device global memory.  NVIDIA GPUs have a few special tricks
                // for optimizing access to constant memory.  Using global memory
                // here would have worked just as well.  See the Programmer's
                // Guide for more information about constant memory.

                GlobalConstants params;

                // Compute Pow2 and Log2 versions of numCircles, this is important for the compaction algorithm,
                // which only works with powers of 2/
                //int powlevel = 0;
                //int temp = numCircles;
                //while(temp >>= 1) powlevel ++;
                //params.numCirclesLog2 = powlevel+1;
                //params.numCirclesPow2 = 1 << (powlevel+1);
                //printf("params.numCirclesPow2 %d\n", 1 << (powlevel + 1));
                params.imageWidth = image->width;
                params.imageHeight = image->height;
                params.bsdfs = deviceBSDFs;
                params.emitters = deviceEmitters;
                params.bvhSubTrees = deviceBVHSubTrees;
                params.triangles = deviceTriangles;
                params.queues1 = deviceRays1;
                params.queues2 = deviceRays2;
                params.intersections = deviceIntersections;
                params.ssImageData = (float4*)deviceSSImageData;
                params.imageData = (float4*)deviceImageData;
                params.qCounts = deviceQueueCounts;
                params.levelIndices = deviceLevelIndices;
                params.c_lookAt = v2f3(c_lookAt);
                params.c_up = v2f3(c_up);
                params.c_origin = v2f3(c_origin);
                params.c_left = v2f3(c_left);
                params.sampleCount = SAMPLES_PER_PIXEL;
                params.minT = deviceMinT;
                params.multiIntersections = deviceMultiIntersections;
                params.intersectionTokens = deviceIntersectionTokens;
                params.randomStates = deviceRandomStates;

                ok = hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));
                if(ok != hipSuccess) {printf("Couldn't allocate memory\n");exit(1);}

                // Also need to copy over the noise lookup tables, so we can
                // implement noise on the GPU
                int* permX;
                int* permY;
                float* value1D;
                //getNoiseTables(&permX, &permY, &value1D);
                //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
                //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
                //hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

                // Copy over the color table that's used by the shading
                // function for circles in the snowflake demo

                /*float lookupTable[COLOR_MAP_SIZE][3] = {
                  {1.f, 1.f, 1.f},
                  {1.f, 1.f, 1.f},
                  {.8f, .9f, 1.f},
                  {.8f, .9f, 1.f},
                  {.8f, 0.8f, 1.f},
                  };

                  hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);*/
                printf("Finished allocation and copy\n");
            }

            // allocOutputImage --
            //
            // Allocate buffer the renderer will render into.  Check status of
            // image first to avoid memory leak.
            void CudaRenderer::allocOutputImage(int width, int height) {
                printf("Image alloc\n");
                if (image)
                    delete image;
                image = new Image(width, height);
                printf("Done Image alloc\n");
            }

            // clearImage --
            //
            // Clear the renderer's target image.  The state of the image after
            // the clear depends on the scene being rendered.
            void CudaRenderer::clearImage() {

                dim3 blockDim(16, 16, 1);
                dim3 gridDim(
                        (image->width + blockDim.x - 1) / blockDim.x,
                        (image->height + blockDim.y - 1) / blockDim.y);

                kernelClearImage<<<gridDim, blockDim>>>(0.f, 0.f, 0.f, 0.f);

                hipDeviceSynchronize();
                printf("Done cleaning\n");
            }

            void CudaRenderer::clearIntersections() {
                int numRays = image->height * image->width * SAMPLES_PER_PIXEL;
                int threadCount = 1024;

                dim3 blockDim(threadCount, 1);
                dim3 gridDim(numRays / threadCount, 1);

                kernelClearIntersections<<<gridDim, blockDim>>>();
            }
            
            void CudaRenderer::render() {

                //printf("Started rendering %d\n", batchSize);fflush(stdout);
                // 256 threads per block is a healthy number
                //dim3 blockDim(NUM_THREADS, 1);
                //dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);
                //dim3 gridDim(numCircles);

                //dim3 blockDim(1024);
                //dim3 gridDim(1024 * (NUM_CIRCLES_PER_BLOCK >> 10));
                dim3 primaryRaysBlockDim(32, 32);
                dim3 primaryRaysGridDim(image->width >> 5, image->height >> 5);

                int blocksPerNode = (image->width * image->height * SAMPLES_PER_PIXEL) / RAYS_PER_BLOCK;
                dim3 rayIntersectBlockDim(RAYS_PER_BLOCK, 1);
                dim3 rayIntersectGridDim(blocksPerNode, 1);
                
                int iidBlocksPerNode = (image->width * image->height * SAMPLES_PER_PIXEL) / 1024;
                dim3 intersectionBlockDim(1024, 1);
                dim3 intersectionGridDim(iidBlocksPerNode, 1);

                int imageBlocksPerNode = (image->width * image->height) / 1024;
                dim3 imageBlockDim(1024, 1);
                dim3 imageGridDim(imageBlocksPerNode, 1);
                
                int a = 0;
                for(int i = 0; i < levelCounts.size(); i++)
                    a += levelCounts[i];
                
                //dim3 queueCountsBlockDim(400, 1);
                //dim3 queueCountsGridDim(1, 1);
                
                kernelSetupRandomSeeds<<<intersectionGridDim, intersectionBlockDim>>>();       
                
                hipDeviceSynchronize();
                //double start = CycleTimer::currentSeconds();
                kernelPrimaryRays<<<primaryRaysGridDim, primaryRaysBlockDim>>>();

                hipDeviceSynchronize();
                // TODO: Concurrent execution.
                clearIntersections();

                hipDeviceSynchronize();
                
                printf("Reset counts: %d\n", a);
                kernelResetCounts<<<1, a>>>();

                hipDeviceSynchronize();
                
                kernelScanCounts<<<1,levelCounts[0]>>>(0);

                hipDeviceSynchronize();

                kernelRayIntersectSingle<<<rayIntersectGridDim, rayIntersectBlockDim>>>(0);

                hipDeviceSynchronize();
                    
                // Compute level indices.
                for(int level = 1; level < levelCounts.size(); level ++) {
                    //if(level != levelCounts.size() - 1) {
                    printf("kernelPrintLevelLists\n");
                    kernelPrintLevelLists<<<1,1>>>(level, levelCounts[level]);
                    hipDeviceSynchronize();
                    //}
                    // for(int level = 1; level < 2; level++) {
                    int totalCount = image->height * image->width * SAMPLES_PER_PIXEL * levelCounts[level];
                    int numBlocks = totalCount / RAYS_PER_BLOCK;
                    //int numBlocks = totalCount / RAYS_PER_BLOCK;
                    printf("kernelIntersectLevel: %d, %d, %d, %d\n", totalCount, numBlocks, levelCounts[level], SAMPLES_PER_PIXEL * image->height * image->width); 
                    
                    
                    kernelScanCounts<<<1,levelCounts[level]>>>(level);
                    
                    hipDeviceSynchronize();

                    dim3 rayIntersectLevelBlockDim(RAYS_PER_BLOCK, 1);
                    dim3 rayIntersectLevelGridDim(numBlocks, 1);
                    kernelRayIntersectLevel<<<numBlocks, RAYS_PER_BLOCK>>>(level);
                    hipDeviceSynchronize();

                }

                //hipDeviceSynchronize();

                //kernelPrintQueueCounts<<<queueCountsGridDim, queueCountsBlockDim>>>();

                //hipDeviceSynchronize();
                
                //printf("kernelMergeIntersections\n");
                kernelMergeIntersections<<<intersectionGridDim, intersectionBlockDim>>>();
                
                hipDeviceSynchronize();
                
                //printf("kernelUpdateSSImage\n");

                kernelDirectLightRays<<<intersectionGridDim, intersectionBlockDim>>>();

                hipDeviceSynchronize();
                
                clearIntersections();

                
                hipDeviceSynchronize();
                      
                // BOUNCE TWO (DIRECT LIGHT) 
                
                printf("Reset counts: %d\n", a);
                kernelResetCounts<<<1, a>>>();

                hipDeviceSynchronize();
                
                kernelScanCounts<<<1,levelCounts[0]>>>(0);

                hipDeviceSynchronize();

                kernelRayIntersectSingle<<<rayIntersectGridDim, rayIntersectBlockDim>>>(0);

                hipDeviceSynchronize();

                // Compute level indices.
                for(int level = 1; level < levelCounts.size(); level ++) {
                    //if(level != levelCounts.size() - 1) {
                    printf("kernelPrintLevelLists DLIGHT %d\n", level);
                    kernelPrintLevelLists<<<1,1>>>(level, levelCounts[level]);
                    hipDeviceSynchronize();
                    //}
                    // for(int level = 1; level < 2; level++) {
                    int totalCount = image->height * image->width * SAMPLES_PER_PIXEL * levelCounts[level];
                    int numBlocks = totalCount / RAYS_PER_BLOCK;
                    //int numBlocks = totalCount / RAYS_PER_BLOCK;
                    printf("kernelIntersectLevel: %d, %d, %d, %d\n", totalCount, numBlocks, levelCounts[level], SAMPLES_PER_PIXEL * image->height * image->width); 
                    
                    kernelScanCounts<<<1,levelCounts[level]>>>(level);
                    
                    hipDeviceSynchronize();

                    dim3 rayIntersectLevelBlockDim(RAYS_PER_BLOCK, 1);
                    dim3 rayIntersectLevelGridDim(numBlocks, 1);
                    kernelRayIntersectLevel<<<numBlocks, RAYS_PER_BLOCK>>>(level);
                    hipDeviceSynchronize();
                }


                //hipDeviceSynchronize();

                //kernelPrintQueueCounts<<<queueCountsGridDim, queueCountsBlockDim>>>();

                //hipDeviceSynchronize();
                
                //printf("kernelMergeIntersections\n");
                kernelMergeIntersections<<<intersectionGridDim, intersectionBlockDim>>>();
                
                hipDeviceSynchronize();
                
                kernelUpdateSSImage<<<intersectionGridDim, intersectionBlockDim>>>();
                
                hipDeviceSynchronize();
                
                //printf("kernelReconstructImage\n");
                kernelReconstructImage<<<imageGridDim, imageBlockDim>>>();
                
                hipDeviceSynchronize();
                
                //kernelDirectLightRays<<<intersectionGridDim, intersectionBlockDim>>>();

                //hipDeviceSynchronize();



                //hipDeviceSynchronize();

                
                }

            }
